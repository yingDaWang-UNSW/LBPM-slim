/*
  Copyright 2013--2018 James E. McClure, Virginia Polytechnic & State University

  This file is part of the Open Porous Media project (OPM).
  OPM is free software: you can redistribute it and/or modify
  it under the terms of the GNU General Public License as published by
  the Free Software Foundation, either version 3 of the License, or
  (at your option) any later version.
  OPM is distributed in the hope that it will be useful,
  but WITHOUT ANY WARRANTY; without even the implied warranty of
  MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the
  GNU General Public License for more details.
  You should have received a copy of the GNU General Public License
  along with OPM.  If not, see <http://www.gnu.org/licenses/>.
*/

#include <hip/hip_runtime.h>
#include <math.h>
#include <stdio.h>
#include <hip/hip_runtime_api.h>

#define NBLOCKS 1024
#define NTHREADS 256

__global__  void dvc_ScaLBL_Color_Init(char *ID, double *Den, double *Phi, double das, double dbs, int Nx, int Ny, int Nz)
{
	//int i,j,k;
	int n,N;
	char id;
	N = Nx*Ny*Nz;

	int S = N/NBLOCKS/NTHREADS + 1;
	for (int s=0; s<S; s++){
		//........Get 1-D index for this thread....................
		n = S*blockIdx.x*blockDim.x + s*blockDim.x + threadIdx.x;
		if (n<N){
		
  		id=ID[n];	
 		//.......Back out the 3-D indices for node n..............
		//k = n/(Nx*Ny);
		//j = (n-Nx*Ny*k)/Nx;
		//i = n-Nx*Ny*k-Nx*j;

		if ( id == 1){
			Den[n] = 1.0;
			Den[N+n] = 0.0;
			Phi[n] = 1.0;
		}
		else if ( id == 2){
			Den[n] = 0.0;
			Den[N+n] = 1.0;
			Phi[n] = -1.0;
		}
		else{
			Den[n] = das;
			Den[N+n] = dbs;
			Phi[n] = (das-dbs)/(das+dbs);
		}
		}
	}
}

__global__  void dvc_ScaLBL_Color_BC(int *list, int *Map, double *Phi, double *Den, double vA, double vB, int count, int Np)
{
	int idx,n,nm;
	// Fill the outlet with component b
	idx = blockIdx.x*blockDim.x + threadIdx.x;
	if (idx < count){
		n = list[idx];
		Den[n] = vA;
		Den[Np+n] = vB;
		
		nm = Map[n];
		Phi[nm] = (vA-vB)/(vA+vB);


	}
}

//*************************************************************************


__global__  void dvc_ScaLBL_SetSlice_z(double *Phi, double value, int Nx, int Ny, int Nz, int Slice)
{
	int n = Slice*Nx*Ny +  blockIdx.x*blockDim.x + threadIdx.x;
	if (n < (Slice+1)*Nx*Ny){
		Phi[n] = value;
	}
}

__global__  void dvc_ScaLBL_CopySlice_z(double *Phi, int Nx, int Ny, int Nz, int Source, int Dest){
	double value;
	int n =  blockIdx.x*blockDim.x + threadIdx.x;
	if (n < Nx*Ny){
		value = Phi[Source*Nx*Ny+n];
		Phi[Dest*Nx*Ny+n] = value;
	}
}

__global__  void dvc_ScaLBL_D3Q19_AAeven_Color(int *Map, double *dist, double *Aq, double *Bq, double *Den, double *Phi,
		double *Velocity, double rhoA, double rhoB, double tauA, double tauB, double alpha, double beta,
		double Fx, double Fy, double Fz, int strideY, int strideZ, int start, int finish, int Np){
	int ijk,nn,n;
	double fq;
	// conserved momemnts
	double rho,jx,jy,jz;
	// non-conserved moments
	double m1,m2,m4,m6,m8,m9,m10,m11,m12,m13,m14,m15,m16,m17,m18;
	double m3,m5,m7;
	double nA,nB; // number density
	double a1,b1,a2,b2,nAB,delta;
	double C,nx,ny,nz; //color gradient magnitude and direction
	double ux,uy,uz;
	double phi,tau,rho0,rlx_setA,rlx_setB;

	const double mrt_V1=0.05263157894736842;
	const double mrt_V2=0.012531328320802;
	const double mrt_V3=0.04761904761904762;
	const double mrt_V4=0.004594820384294068;
	const double mrt_V5=0.01587301587301587;
	const double mrt_V6=0.0555555555555555555555555;
	const double mrt_V7=0.02777777777777778;
	const double mrt_V8=0.08333333333333333;
	const double mrt_V9=0.003341687552213868;
	const double mrt_V10=0.003968253968253968;
	const double mrt_V11=0.01388888888888889;
	const double mrt_V12=0.04166666666666666;

	int S = Np/NBLOCKS/NTHREADS + 1;
	for (int s=0; s<S; s++){
		//........Get 1-D index for this thread....................
		n =  S*blockIdx.x*blockDim.x + s*blockDim.x + threadIdx.x + start;
		if (n<finish) {

			// read the component number densities
			nA = Den[n];
			nB = Den[Np + n];

			// compute phase indicator field
			phi=(nA-nB)/(nA+nB);

			// local density
			rho0=rhoA + 0.5*(1.0-phi)*(rhoB-rhoA);
			// local relaxation time
			tau=tauA + 0.5*(1.0-phi)*(tauB-tauA);
			rlx_setA = 1.f/tau;
			rlx_setB = 8.f*(2.f-rlx_setA)/(8.f-rlx_setA);

			// Get the 1D index based on regular data layout
			ijk = Map[n];
			//					COMPUTE THE COLOR GRADIENT
			//........................................................................
			//.................Read Phase Indicator Values............................
			//........................................................................
			nn = ijk-1;							// neighbor index (get convention)
			m1 = Phi[nn];						// get neighbor for phi - 1
			//........................................................................
			nn = ijk+1;							// neighbor index (get convention)
			m2 = Phi[nn];						// get neighbor for phi - 2
			//........................................................................
			nn = ijk-strideY;							// neighbor index (get convention)
			m3 = Phi[nn];					// get neighbor for phi - 3
			//........................................................................
			nn = ijk+strideY;							// neighbor index (get convention)
			m4 = Phi[nn];					// get neighbor for phi - 4
			//........................................................................
			nn = ijk-strideZ;						// neighbor index (get convention)
			m5 = Phi[nn];					// get neighbor for phi - 5
			//........................................................................
			nn = ijk+strideZ;						// neighbor index (get convention)
			m6 = Phi[nn];					// get neighbor for phi - 6
			//........................................................................
			nn = ijk-strideY-1;						// neighbor index (get convention)
			m7 = Phi[nn];					// get neighbor for phi - 7
			//........................................................................
			nn = ijk+strideY+1;						// neighbor index (get convention)
			m8 = Phi[nn];					// get neighbor for phi - 8
			//........................................................................
			nn = ijk+strideY-1;						// neighbor index (get convention)
			m9 = Phi[nn];					// get neighbor for phi - 9
			//........................................................................
			nn = ijk-strideY+1;						// neighbor index (get convention)
			m10 = Phi[nn];					// get neighbor for phi - 10
			//........................................................................
			nn = ijk-strideZ-1;						// neighbor index (get convention)
			m11 = Phi[nn];					// get neighbor for phi - 11
			//........................................................................
			nn = ijk+strideZ+1;						// neighbor index (get convention)
			m12 = Phi[nn];					// get neighbor for phi - 12
			//........................................................................
			nn = ijk+strideZ-1;						// neighbor index (get convention)
			m13 = Phi[nn];					// get neighbor for phi - 13
			//........................................................................
			nn = ijk-strideZ+1;						// neighbor index (get convention)
			m14 = Phi[nn];					// get neighbor for phi - 14
			//........................................................................
			nn = ijk-strideZ-strideY;					// neighbor index (get convention)
			m15 = Phi[nn];					// get neighbor for phi - 15
			//........................................................................
			nn = ijk+strideZ+strideY;					// neighbor index (get convention)
			m16 = Phi[nn];					// get neighbor for phi - 16
			//........................................................................
			nn = ijk+strideZ-strideY;					// neighbor index (get convention)
			m17 = Phi[nn];					// get neighbor for phi - 17
			//........................................................................
			nn = ijk-strideZ+strideY;					// neighbor index (get convention)
			m18 = Phi[nn];					// get neighbor for phi - 18
			//............Compute the Color Gradient...................................
			nx = -(m1-m2+0.5*(m7-m8+m9-m10+m11-m12+m13-m14));
			ny = -(m3-m4+0.5*(m7-m8-m9+m10+m15-m16+m17-m18));
			nz = -(m5-m6+0.5*(m11-m12-m13+m14+m15-m16-m17+m18));

			//...........Normalize the Color Gradient.................................
			C = sqrt(nx*nx+ny*ny+nz*nz);
			double ColorMag = C;
			if (C==0.0) ColorMag=1.0;
			nx = nx/ColorMag;
			ny = ny/ColorMag;
			nz = nz/ColorMag;		

			// q=0
			fq = dist[n];
			rho = fq;
			m1  = -30.0*fq;
			m2  = 12.0*fq;

			// q=1
			fq = dist[2*Np+n];
			rho += fq;
			m1 -= 11.0*fq;
			m2 -= 4.0*fq;
			jx = fq;
			m4 = -4.0*fq;
			m9 = 2.0*fq;
			m10 = -4.0*fq;

			// f2 = dist[10*Np+n];
			fq = dist[1*Np+n];
			rho += fq;
			m1 -= 11.0*(fq);
			m2 -= 4.0*(fq);
			jx -= fq;
			m4 += 4.0*(fq);
			m9 += 2.0*(fq);
			m10 -= 4.0*(fq);

			// q=3
			fq = dist[4*Np+n];
			rho += fq;
			m1 -= 11.0*fq;
			m2 -= 4.0*fq;
			jy = fq;
			m6 = -4.0*fq;
			m9 -= fq;
			m10 += 2.0*fq;
			m11 = fq;
			m12 = -2.0*fq;

			// q = 4
			fq = dist[3*Np+n];
			rho+= fq;
			m1 -= 11.0*fq;
			m2 -= 4.0*fq;
			jy -= fq;
			m6 += 4.0*fq;
			m9 -= fq;
			m10 += 2.0*fq;
			m11 += fq;
			m12 -= 2.0*fq;

			// q=5
			fq = dist[6*Np+n];
			rho += fq;
			m1 -= 11.0*fq;
			m2 -= 4.0*fq;
			jz = fq;
			m8 = -4.0*fq;
			m9 -= fq;
			m10 += 2.0*fq;
			m11 -= fq;
			m12 += 2.0*fq;

			// q = 6
			fq = dist[5*Np+n];
			rho+= fq;
			m1 -= 11.0*fq;
			m2 -= 4.0*fq;
			jz -= fq;
			m8 += 4.0*fq;
			m9 -= fq;
			m10 += 2.0*fq;
			m11 -= fq;
			m12 += 2.0*fq;

			// q=7
			fq = dist[8*Np+n];
			rho += fq;
			m1 += 8.0*fq;
			m2 += fq;
			jx += fq;
			m4 += fq;
			jy += fq;
			m6 += fq;
			m9  += fq;
			m10 += fq;
			m11 += fq;
			m12 += fq;
			m13 = fq;
			m16 = fq;
			m17 = -fq;

			// q = 8
			fq = dist[7*Np+n];
			rho += fq;
			m1 += 8.0*fq;
			m2 += fq;
			jx -= fq;
			m4 -= fq;
			jy -= fq;
			m6 -= fq;
			m9 += fq;
			m10 += fq;
			m11 += fq;
			m12 += fq;
			m13 += fq;
			m16 -= fq;
			m17 += fq;

			// q=9
			fq = dist[10*Np+n];
			rho += fq;
			m1 += 8.0*fq;
			m2 += fq;
			jx += fq;
			m4 += fq;
			jy -= fq;
			m6 -= fq;
			m9 += fq;
			m10 += fq;
			m11 += fq;
			m12 += fq;
			m13 -= fq;
			m16 += fq;
			m17 += fq;

			// q = 10
			fq = dist[9*Np+n];
			rho += fq;
			m1 += 8.0*fq;
			m2 += fq;
			jx -= fq;
			m4 -= fq;
			jy += fq;
			m6 += fq;
			m9 += fq;
			m10 += fq;
			m11 += fq;
			m12 += fq;
			m13 -= fq;
			m16 -= fq;
			m17 -= fq;

			// q=11
			fq = dist[12*Np+n];
			rho += fq;
			m1 += 8.0*fq;
			m2 += fq;
			jx += fq;
			m4 += fq;
			jz += fq;
			m8 += fq;
			m9 += fq;
			m10 += fq;
			m11 -= fq;
			m12 -= fq;
			m15 = fq;
			m16 -= fq;
			m18 = fq;

			// q=12
			fq = dist[11*Np+n];
			rho += fq;
			m1 += 8.0*fq;
			m2 += fq;
			jx -= fq;
			m4 -= fq;
			jz -= fq;
			m8 -= fq;
			m9 += fq;
			m10 += fq;
			m11 -= fq;
			m12 -= fq;
			m15 += fq;
			m16 += fq;
			m18 -= fq;

			// q=13
			fq = dist[14*Np+n];
			rho += fq;
			m1 += 8.0*fq;
			m2 += fq;
			jx += fq;
			m4 += fq;
			jz -= fq;
			m8 -= fq;
			m9 += fq;
			m10 += fq;
			m11 -= fq;
			m12 -= fq;
			m15 -= fq;
			m16 -= fq;
			m18 -= fq;

			// q=14
			fq = dist[13*Np+n];
			rho += fq;
			m1 += 8.0*fq;
			m2 += fq;
			jx -= fq;
			m4 -= fq;
			jz += fq;
			m8 += fq;
			m9 += fq;
			m10 += fq;
			m11 -= fq;
			m12 -= fq;
			m15 -= fq;
			m16 += fq;
			m18 += fq;

			// q=15
			fq = dist[16*Np+n];
			rho += fq;
			m1 += 8.0*fq;
			m2 += fq;
			jy += fq;
			m6 += fq;
			jz += fq;
			m8 += fq;
			m9 -= 2.0*fq;
			m10 -= 2.0*fq;
			m14 = fq;
			m17 += fq;
			m18 -= fq;

			// q=16
			fq = dist[15*Np+n];
			rho += fq;
			m1 += 8.0*fq;
			m2 += fq;
			jy -= fq;
			m6 -= fq;
			jz -= fq;
			m8 -= fq;
			m9 -= 2.0*fq;
			m10 -= 2.0*fq;
			m14 += fq;
			m17 -= fq;
			m18 += fq;

			// q=17
			fq = dist[18*Np+n];
			rho += fq;
			m1 += 8.0*fq;
			m2 += fq;
			jy += fq;
			m6 += fq;
			jz -= fq;
			m8 -= fq;
			m9 -= 2.0*fq;
			m10 -= 2.0*fq;
			m14 -= fq;
			m17 += fq;
			m18 += fq;

			// q=18
			fq = dist[17*Np+n];
			rho += fq;
			m1 += 8.0*fq;
			m2 += fq;
			jy -= fq;
			m6 -= fq;
			jz += fq;
			m8 += fq;
			m9 -= 2.0*fq;
			m10 -= 2.0*fq;
			m14 -= fq;
			m17 -= fq;
			m18 -= fq;

			//........................................................................
			//..............carry out relaxation process..............................
			//..........Toelke, Fruediger et. al. 2006................................
			if (C == 0.0)	nx = ny = nz = 0.0;
			m1 = m1 + rlx_setA*((19*(jx*jx+jy*jy+jz*jz)/rho0 - 11*rho) -19*alpha*C - m1);
			m2 = m2 + rlx_setA*((3*rho - 5.5*(jx*jx+jy*jy+jz*jz)/rho0)- m2);
			m4 = m4 + rlx_setB*((-0.6666666666666666*jx)- m4);
			m6 = m6 + rlx_setB*((-0.6666666666666666*jy)- m6);
			m8 = m8 + rlx_setB*((-0.6666666666666666*jz)- m8);
			m9 = m9 + rlx_setA*(((2*jx*jx-jy*jy-jz*jz)/rho0) + 0.5*alpha*C*(2*nx*nx-ny*ny-nz*nz) - m9);
			m10 = m10 + rlx_setA*( - m10);
			m11 = m11 + rlx_setA*(((jy*jy-jz*jz)/rho0) + 0.5*alpha*C*(ny*ny-nz*nz)- m11);
			m12 = m12 + rlx_setA*( - m12);
			m13 = m13 + rlx_setA*( (jx*jy/rho0) + 0.5*alpha*C*nx*ny - m13);
			m14 = m14 + rlx_setA*( (jy*jz/rho0) + 0.5*alpha*C*ny*nz - m14);
			m15 = m15 + rlx_setA*( (jx*jz/rho0) + 0.5*alpha*C*nx*nz - m15);
			m16 = m16 + rlx_setB*( - m16);
			m17 = m17 + rlx_setB*( - m17);
			m18 = m18 + rlx_setB*( - m18);

			//.......................................................................................................
			//.................inverse transformation......................................................

			// q=0
			fq = mrt_V1*rho-mrt_V2*m1+mrt_V3*m2;
			dist[n] = fq;

			// q = 1
			fq = mrt_V1*rho-mrt_V4*m1-mrt_V5*m2+0.1*(jx-m4)+mrt_V6*(m9-m10) + 0.16666666*Fx;
			dist[1*Np+n] = fq;

			// q=2
			fq = mrt_V1*rho-mrt_V4*m1-mrt_V5*m2+0.1*(m4-jx)+mrt_V6*(m9-m10) -  0.16666666*Fx;
			dist[2*Np+n] = fq;

			// q = 3
			fq = mrt_V1*rho-mrt_V4*m1-mrt_V5*m2+0.1*(jy-m6)+mrt_V7*(m10-m9)+mrt_V8*(m11-m12) + 0.16666666*Fy;
			dist[3*Np+n] = fq;

			// q = 4
			fq = mrt_V1*rho-mrt_V4*m1-mrt_V5*m2+0.1*(m6-jy)+mrt_V7*(m10-m9)+mrt_V8*(m11-m12) - 0.16666666*Fy;
			dist[4*Np+n] = fq;

			// q = 5
			fq = mrt_V1*rho-mrt_V4*m1-mrt_V5*m2+0.1*(jz-m8)+mrt_V7*(m10-m9)+mrt_V8*(m12-m11) + 0.16666666*Fz;
			dist[5*Np+n] = fq;

			// q = 6
			fq = mrt_V1*rho-mrt_V4*m1-mrt_V5*m2+0.1*(m8-jz)+mrt_V7*(m10-m9)+mrt_V8*(m12-m11) - 0.16666666*Fz;
			dist[6*Np+n] = fq;

			// q = 7
			fq = mrt_V1*rho+mrt_V9*m1+mrt_V10*m2+0.1*(jx+jy)+0.025*(m4+m6)+
					mrt_V7*m9+mrt_V11*m10+mrt_V8*m11+mrt_V12*m12+0.25*m13+0.125*(m16-m17) + 0.08333333333*(Fx+Fy);
			dist[7*Np+n] = fq;


			// q = 8
			fq = mrt_V1*rho+mrt_V9*m1+mrt_V10*m2-0.1*(jx+jy)-0.025*(m4+m6) +mrt_V7*m9+mrt_V11*m10+mrt_V8*m11
					+mrt_V12*m12+0.25*m13+0.125*(m17-m16) - 0.08333333333*(Fx+Fy);
			dist[8*Np+n] = fq;

			// q = 9
			fq = mrt_V1*rho+mrt_V9*m1+mrt_V10*m2+0.1*(jx-jy)+0.025*(m4-m6)+
					mrt_V7*m9+mrt_V11*m10+mrt_V8*m11+mrt_V12*m12-0.25*m13+0.125*(m16+m17) + 0.08333333333*(Fx-Fy);
			dist[9*Np+n] = fq;

			// q = 10
			fq = mrt_V1*rho+mrt_V9*m1+mrt_V10*m2+0.1*(jy-jx)+0.025*(m6-m4)+
					mrt_V7*m9+mrt_V11*m10+mrt_V8*m11+mrt_V12*m12-0.25*m13-0.125*(m16+m17)- 0.08333333333*(Fx-Fy);
			dist[10*Np+n] = fq;


			// q = 11
			fq = mrt_V1*rho+mrt_V9*m1
					+mrt_V10*m2+0.1*(jx+jz)+0.025*(m4+m8)
					+mrt_V7*m9+mrt_V11*m10-mrt_V8*m11
					-mrt_V12*m12+0.25*m15+0.125*(m18-m16) + 0.08333333333*(Fx+Fz);
			dist[11*Np+n] = fq;

			// q = 12
			fq = mrt_V1*rho+mrt_V9*m1+mrt_V10*m2-0.1*(jx+jz)-0.025*(m4+m8)+
					mrt_V7*m9+mrt_V11*m10-mrt_V8*m11-mrt_V12*m12+0.25*m15+0.125*(m16-m18)-0.08333333333*(Fx+Fz);
			dist[12*Np+n] = fq;

			// q = 13
			fq = mrt_V1*rho+mrt_V9*m1
					+mrt_V10*m2+0.1*(jx-jz)+0.025*(m4-m8)
					+mrt_V7*m9+mrt_V11*m10-mrt_V8*m11
					-mrt_V12*m12-0.25*m15-0.125*(m16+m18) + 0.08333333333*(Fx-Fz);
			dist[13*Np+n] = fq;

			// q= 14
			fq = mrt_V1*rho+mrt_V9*m1
					+mrt_V10*m2+0.1*(jz-jx)+0.025*(m8-m4)
					+mrt_V7*m9+mrt_V11*m10-mrt_V8*m11
					-mrt_V12*m12-0.25*m15+0.125*(m16+m18) - 0.08333333333*(Fx-Fz);

			dist[14*Np+n] = fq;

			// q = 15
			fq = mrt_V1*rho+mrt_V9*m1
					+mrt_V10*m2+0.1*(jy+jz)+0.025*(m6+m8)
					-mrt_V6*m9-mrt_V7*m10+0.25*m14+0.125*(m17-m18) + 0.08333333333*(Fy+Fz);
			dist[15*Np+n] = fq;

			// q = 16
			fq =  mrt_V1*rho+mrt_V9*m1
					+mrt_V10*m2-0.1*(jy+jz)-0.025*(m6+m8)
					-mrt_V6*m9-mrt_V7*m10+0.25*m14+0.125*(m18-m17)- 0.08333333333*(Fy+Fz);
			dist[16*Np+n] = fq;


			// q = 17
			fq = mrt_V1*rho+mrt_V9*m1
					+mrt_V10*m2+0.1*(jy-jz)+0.025*(m6-m8)
					-mrt_V6*m9-mrt_V7*m10-0.25*m14+0.125*(m17+m18) + 0.08333333333*(Fy-Fz);
			dist[17*Np+n] = fq;

			// q = 18
			fq = mrt_V1*rho+mrt_V9*m1
					+mrt_V10*m2+0.1*(jz-jy)+0.025*(m8-m6)
					-mrt_V6*m9-mrt_V7*m10-0.25*m14-0.125*(m17+m18) - 0.08333333333*(Fy-Fz);
			dist[18*Np+n] = fq;

			//........................................................................

			// write the velocity 
			ux = jx / rho0;
			uy = jy / rho0;
			uz = jz / rho0;
			Velocity[n] = ux;
			Velocity[Np+n] = uy;
			Velocity[2*Np+n] = uz;

			// Instantiate mass transport distributions
			// Stationary value - distribution 0

			nAB = 1.0/(nA+nB);
			Aq[n] = 0.3333333333333333*nA;
			Bq[n] = 0.3333333333333333*nB;

			//...............................................
			// q = 0,2,4
			// Cq = {1,0,0}, {0,1,0}, {0,0,1}
			delta = beta*nA*nB*nAB*0.1111111111111111*nx;
			if (!(nA*nB*nAB>0)) delta=0;
			a1 = nA*(0.1111111111111111*(1+4.5*ux))+delta;
			b1 = nB*(0.1111111111111111*(1+4.5*ux))-delta;
			a2 = nA*(0.1111111111111111*(1-4.5*ux))-delta;
			b2 = nB*(0.1111111111111111*(1-4.5*ux))+delta;

			Aq[1*Np+n] = a1;
			Bq[1*Np+n] = b1;
			Aq[2*Np+n] = a2;
			Bq[2*Np+n] = b2;

			//...............................................
			// q = 2
			// Cq = {0,1,0}
			delta = beta*nA*nB*nAB*0.1111111111111111*ny;
			if (!(nA*nB*nAB>0)) delta=0;
			a1 = nA*(0.1111111111111111*(1+4.5*uy))+delta;
			b1 = nB*(0.1111111111111111*(1+4.5*uy))-delta;
			a2 = nA*(0.1111111111111111*(1-4.5*uy))-delta;
			b2 = nB*(0.1111111111111111*(1-4.5*uy))+delta;

			Aq[3*Np+n] = a1;
			Bq[3*Np+n] = b1;
			Aq[4*Np+n] = a2;
			Bq[4*Np+n] = b2;
			//...............................................
			// q = 4
			// Cq = {0,0,1}
			delta = beta*nA*nB*nAB*0.1111111111111111*nz;
			if (!(nA*nB*nAB>0)) delta=0;
			a1 = nA*(0.1111111111111111*(1+4.5*uz))+delta;
			b1 = nB*(0.1111111111111111*(1+4.5*uz))-delta;
			a2 = nA*(0.1111111111111111*(1-4.5*uz))-delta;
			b2 = nB*(0.1111111111111111*(1-4.5*uz))+delta;

			Aq[5*Np+n] = a1;
			Bq[5*Np+n] = b1;
			Aq[6*Np+n] = a2;
			Bq[6*Np+n] = b2;
			//...............................................

		}
	}
}


__global__ void dvc_ScaLBL_D3Q19_AAodd_Color(int *neighborList, int *Map, double *dist, double *Aq, double *Bq, double *Den,
		 double *Phi, double *Velocity, double rhoA, double rhoB, double tauA, double tauB, double alpha, double beta,
		double Fx, double Fy, double Fz, int strideY, int strideZ, int start, int finish, int Np){

	int n,nn,ijk,nread;
	int nr1,nr2,nr3,nr4,nr5,nr6;
	int nr7,nr8,nr9,nr10;
	int nr11,nr12,nr13,nr14;
	//int nr15,nr16,nr17,nr18;
	double fq;
	// conserved momemnts
	double rho,jx,jy,jz;
	// non-conserved moments
	double m1,m2,m4,m6,m8,m9,m10,m11,m12,m13,m14,m15,m16,m17,m18;
	double m3,m5,m7;
	double nA,nB; // number density
	double a1,b1,a2,b2,nAB,delta;
	double C,nx,ny,nz; //color gradient magnitude and direction
	double ux,uy,uz;
	double phi,tau,rho0,rlx_setA,rlx_setB;

	const double mrt_V1=0.05263157894736842;
	const double mrt_V2=0.012531328320802;
	const double mrt_V3=0.04761904761904762;
	const double mrt_V4=0.004594820384294068;
	const double mrt_V5=0.01587301587301587;
	const double mrt_V6=0.0555555555555555555555555;
	const double mrt_V7=0.02777777777777778;
	const double mrt_V8=0.08333333333333333;
	const double mrt_V9=0.003341687552213868;
	const double mrt_V10=0.003968253968253968;
	const double mrt_V11=0.01388888888888889;
	const double mrt_V12=0.04166666666666666;

	int S = Np/NBLOCKS/NTHREADS + 1;
	for (int s=0; s<S; s++){
		//........Get 1-D index for this thread....................
		n =  S*blockIdx.x*blockDim.x + s*blockDim.x + threadIdx.x + start;
		if (n<finish) {
			// read the component number densities
			nA = Den[n];
			nB = Den[Np + n];

			// compute phase indicator field
			phi=(nA-nB)/(nA+nB);

			// local density
			rho0=rhoA + 0.5*(1.0-phi)*(rhoB-rhoA);
			// local relaxation time
			tau=tauA + 0.5*(1.0-phi)*(tauB-tauA);
			rlx_setA = 1.f/tau;
			rlx_setB = 8.f*(2.f-rlx_setA)/(8.f-rlx_setA);
			
			// Get the 1D index based on regular data layout
			ijk = Map[n];
			//					COMPUTE THE COLOR GRADIENT
			//........................................................................
			//.................Read Phase Indicator Values............................
			//........................................................................
			nn = ijk-1;							// neighbor index (get convention)
			m1 = Phi[nn];						// get neighbor for phi - 1
			//........................................................................
			nn = ijk+1;							// neighbor index (get convention)
			m2 = Phi[nn];						// get neighbor for phi - 2
			//........................................................................
			nn = ijk-strideY;							// neighbor index (get convention)
			m3 = Phi[nn];					// get neighbor for phi - 3
			//........................................................................
			nn = ijk+strideY;							// neighbor index (get convention)
			m4 = Phi[nn];					// get neighbor for phi - 4
			//........................................................................
			nn = ijk-strideZ;						// neighbor index (get convention)
			m5 = Phi[nn];					// get neighbor for phi - 5
			//........................................................................
			nn = ijk+strideZ;						// neighbor index (get convention)
			m6 = Phi[nn];					// get neighbor for phi - 6
			//........................................................................
			nn = ijk-strideY-1;						// neighbor index (get convention)
			m7 = Phi[nn];					// get neighbor for phi - 7
			//........................................................................
			nn = ijk+strideY+1;						// neighbor index (get convention)
			m8 = Phi[nn];					// get neighbor for phi - 8
			//........................................................................
			nn = ijk+strideY-1;						// neighbor index (get convention)
			m9 = Phi[nn];					// get neighbor for phi - 9
			//........................................................................
			nn = ijk-strideY+1;						// neighbor index (get convention)
			m10 = Phi[nn];					// get neighbor for phi - 10
			//........................................................................
			nn = ijk-strideZ-1;						// neighbor index (get convention)
			m11 = Phi[nn];					// get neighbor for phi - 11
			//........................................................................
			nn = ijk+strideZ+1;						// neighbor index (get convention)
			m12 = Phi[nn];					// get neighbor for phi - 12
			//........................................................................
			nn = ijk+strideZ-1;						// neighbor index (get convention)
			m13 = Phi[nn];					// get neighbor for phi - 13
			//........................................................................
			nn = ijk-strideZ+1;						// neighbor index (get convention)
			m14 = Phi[nn];					// get neighbor for phi - 14
			//........................................................................
			nn = ijk-strideZ-strideY;					// neighbor index (get convention)
			m15 = Phi[nn];					// get neighbor for phi - 15
			//........................................................................
			nn = ijk+strideZ+strideY;					// neighbor index (get convention)
			m16 = Phi[nn];					// get neighbor for phi - 16
			//........................................................................
			nn = ijk+strideZ-strideY;					// neighbor index (get convention)
			m17 = Phi[nn];					// get neighbor for phi - 17
			//........................................................................
			nn = ijk-strideZ+strideY;					// neighbor index (get convention)
			m18 = Phi[nn];					// get neighbor for phi - 18
			//............Compute the Color Gradient...................................
			nx = -(m1-m2+0.5*(m7-m8+m9-m10+m11-m12+m13-m14));
			ny = -(m3-m4+0.5*(m7-m8-m9+m10+m15-m16+m17-m18));
			nz = -(m5-m6+0.5*(m11-m12-m13+m14+m15-m16-m17+m18));

			//...........Normalize the Color Gradient.................................
			C = sqrt(nx*nx+ny*ny+nz*nz);
			if (C==0.0) C=1.0;
			nx = nx/C;
			ny = ny/C;
			nz = nz/C;		

			// q=0
			fq = dist[n];
			rho = fq;
			m1  = -30.0*fq;
			m2  = 12.0*fq;

			// q=1
			//nread = neighborList[n]; // neighbor 2 
			//fq = dist[nread]; // reading the f1 data into register fq		
			nr1 = neighborList[n]; 
			fq = dist[nr1]; // reading the f1 data into register fq
			rho += fq;
			m1 -= 11.0*fq;
			m2 -= 4.0*fq;
			jx = fq;
			m4 = -4.0*fq;
			m9 = 2.0*fq;
			m10 = -4.0*fq;

			// f2 = dist[10*Np+n];
			//nread = neighborList[n+Np]; // neighbor 1 ( < 10Np => even part of dist)
			//fq = dist[nread];  // reading the f2 data into register fq
			nr2 = neighborList[n+Np]; // neighbor 1 ( < 10Np => even part of dist)
			fq = dist[nr2];  // reading the f2 data into register fq
			rho += fq;
			m1 -= 11.0*(fq);
			m2 -= 4.0*(fq);
			jx -= fq;
			m4 += 4.0*(fq);
			m9 += 2.0*(fq);
			m10 -= 4.0*(fq);

			// q=3
			//nread = neighborList[n+2*Np]; // neighbor 4
			//fq = dist[nread];
			nr3 = neighborList[n+2*Np]; // neighbor 4
			fq = dist[nr3];
			rho += fq;
			m1 -= 11.0*fq;
			m2 -= 4.0*fq;
			jy = fq;
			m6 = -4.0*fq;
			m9 -= fq;
			m10 += 2.0*fq;
			m11 = fq;
			m12 = -2.0*fq;

			// q = 4
			//nread = neighborList[n+3*Np]; // neighbor 3
			//fq = dist[nread];
			nr4 = neighborList[n+3*Np]; // neighbor 3
			fq = dist[nr4];
			rho+= fq;
			m1 -= 11.0*fq;
			m2 -= 4.0*fq;
			jy -= fq;
			m6 += 4.0*fq;
			m9 -= fq;
			m10 += 2.0*fq;
			m11 += fq;
			m12 -= 2.0*fq;

			// q=5
			//nread = neighborList[n+4*Np];
			//fq = dist[nread];
			nr5 = neighborList[n+4*Np];
			fq = dist[nr5];
			rho += fq;
			m1 -= 11.0*fq;
			m2 -= 4.0*fq;
			jz = fq;
			m8 = -4.0*fq;
			m9 -= fq;
			m10 += 2.0*fq;
			m11 -= fq;
			m12 += 2.0*fq;


			// q = 6
			//nread = neighborList[n+5*Np];
			//fq = dist[nread];
			nr6 = neighborList[n+5*Np];
			fq = dist[nr6];
			rho+= fq;
			m1 -= 11.0*fq;
			m2 -= 4.0*fq;
			jz -= fq;
			m8 += 4.0*fq;
			m9 -= fq;
			m10 += 2.0*fq;
			m11 -= fq;
			m12 += 2.0*fq;

			// q=7
			//nread = neighborList[n+6*Np];
			//fq = dist[nread];
			nr7 = neighborList[n+6*Np];
			fq = dist[nr7];
			rho += fq;
			m1 += 8.0*fq;
			m2 += fq;
			jx += fq;
			m4 += fq;
			jy += fq;
			m6 += fq;
			m9  += fq;
			m10 += fq;
			m11 += fq;
			m12 += fq;
			m13 = fq;
			m16 = fq;
			m17 = -fq;

			// q = 8
			//nread = neighborList[n+7*Np];
			//fq = dist[nread];
			nr8 = neighborList[n+7*Np];
			fq = dist[nr8];
			rho += fq;
			m1 += 8.0*fq;
			m2 += fq;
			jx -= fq;
			m4 -= fq;
			jy -= fq;
			m6 -= fq;
			m9 += fq;
			m10 += fq;
			m11 += fq;
			m12 += fq;
			m13 += fq;
			m16 -= fq;
			m17 += fq;

			// q=9
			//nread = neighborList[n+8*Np];
			//fq = dist[nread];
			nr9 = neighborList[n+8*Np];
			fq = dist[nr9];
			rho += fq;
			m1 += 8.0*fq;
			m2 += fq;
			jx += fq;
			m4 += fq;
			jy -= fq;
			m6 -= fq;
			m9 += fq;
			m10 += fq;
			m11 += fq;
			m12 += fq;
			m13 -= fq;
			m16 += fq;
			m17 += fq;

			// q = 10
			//nread = neighborList[n+9*Np];
			//fq = dist[nread];
			nr10 = neighborList[n+9*Np];
			fq = dist[nr10];
			rho += fq;
			m1 += 8.0*fq;
			m2 += fq;
			jx -= fq;
			m4 -= fq;
			jy += fq;
			m6 += fq;
			m9 += fq;
			m10 += fq;
			m11 += fq;
			m12 += fq;
			m13 -= fq;
			m16 -= fq;
			m17 -= fq;

			// q=11
			//nread = neighborList[n+10*Np];
			//fq = dist[nread];
			nr11 = neighborList[n+10*Np];
			fq = dist[nr11];
			rho += fq;
			m1 += 8.0*fq;
			m2 += fq;
			jx += fq;
			m4 += fq;
			jz += fq;
			m8 += fq;
			m9 += fq;
			m10 += fq;
			m11 -= fq;
			m12 -= fq;
			m15 = fq;
			m16 -= fq;
			m18 = fq;

			// q=12
			//nread = neighborList[n+11*Np];
			//fq = dist[nread];
			nr12 = neighborList[n+11*Np];
			fq = dist[nr12];
			rho += fq;
			m1 += 8.0*fq;
			m2 += fq;
			jx -= fq;
			m4 -= fq;
			jz -= fq;
			m8 -= fq;
			m9 += fq;
			m10 += fq;
			m11 -= fq;
			m12 -= fq;
			m15 += fq;
			m16 += fq;
			m18 -= fq;

			// q=13
			//nread = neighborList[n+12*Np];
			//fq = dist[nread];
			nr13 = neighborList[n+12*Np];
			fq = dist[nr13];
			rho += fq;
			m1 += 8.0*fq;
			m2 += fq;
			jx += fq;
			m4 += fq;
			jz -= fq;
			m8 -= fq;
			m9 += fq;
			m10 += fq;
			m11 -= fq;
			m12 -= fq;
			m15 -= fq;
			m16 -= fq;
			m18 -= fq;

			// q=14
			//nread = neighborList[n+13*Np];
			//fq = dist[nread];
			nr14 = neighborList[n+13*Np];
			fq = dist[nr14];
			rho += fq;
			m1 += 8.0*fq;
			m2 += fq;
			jx -= fq;
			m4 -= fq;
			jz += fq;
			m8 += fq;
			m9 += fq;
			m10 += fq;
			m11 -= fq;
			m12 -= fq;
			m15 -= fq;
			m16 += fq;
			m18 += fq;

			// q=15
			nread = neighborList[n+14*Np];
			fq = dist[nread];
			//fq = dist[17*Np+n];
			rho += fq;
			m1 += 8.0*fq;
			m2 += fq;
			jy += fq;
			m6 += fq;
			jz += fq;
			m8 += fq;
			m9 -= 2.0*fq;
			m10 -= 2.0*fq;
			m14 = fq;
			m17 += fq;
			m18 -= fq;

			// q=16
			nread = neighborList[n+15*Np];
			fq = dist[nread];
			//fq = dist[8*Np+n];
			rho += fq;
			m1 += 8.0*fq;
			m2 += fq;
			jy -= fq;
			m6 -= fq;
			jz -= fq;
			m8 -= fq;
			m9 -= 2.0*fq;
			m10 -= 2.0*fq;
			m14 += fq;
			m17 -= fq;
			m18 += fq;

			// q=17
			//fq = dist[18*Np+n];
			nread = neighborList[n+16*Np];
			fq = dist[nread];
			rho += fq;
			m1 += 8.0*fq;
			m2 += fq;
			jy += fq;
			m6 += fq;
			jz -= fq;
			m8 -= fq;
			m9 -= 2.0*fq;
			m10 -= 2.0*fq;
			m14 -= fq;
			m17 += fq;
			m18 += fq;

			// q=18
			nread = neighborList[n+17*Np];
			fq = dist[nread];
			//fq = dist[9*Np+n];
			rho += fq;
			m1 += 8.0*fq;
			m2 += fq;
			jy -= fq;
			m6 -= fq;
			jz += fq;
			m8 += fq;
			m9 -= 2.0*fq;
			m10 -= 2.0*fq;
			m14 -= fq;
			m17 -= fq;
			m18 -= fq;
			
			//........................................................................
			//..............carry out relaxation process..............................
			//..........Toelke, Fruediger et. al. 2006................................
			if (C == 0.0)	nx = ny = nz = 0.0;
			m1 = m1 + rlx_setA*((19*(jx*jx+jy*jy+jz*jz)/rho0 - 11*rho) -19*alpha*C - m1);
			m2 = m2 + rlx_setA*((3*rho - 5.5*(jx*jx+jy*jy+jz*jz)/rho0)- m2);
			m4 = m4 + rlx_setB*((-0.6666666666666666*jx)- m4);
			m6 = m6 + rlx_setB*((-0.6666666666666666*jy)- m6);
			m8 = m8 + rlx_setB*((-0.6666666666666666*jz)- m8);
			m9 = m9 + rlx_setA*(((2*jx*jx-jy*jy-jz*jz)/rho0) + 0.5*alpha*C*(2*nx*nx-ny*ny-nz*nz) - m9);
			m10 = m10 + rlx_setA*( - m10);
			m11 = m11 + rlx_setA*(((jy*jy-jz*jz)/rho0) + 0.5*alpha*C*(ny*ny-nz*nz)- m11);
			m12 = m12 + rlx_setA*( - m12);
			m13 = m13 + rlx_setA*( (jx*jy/rho0) + 0.5*alpha*C*nx*ny - m13);
			m14 = m14 + rlx_setA*( (jy*jz/rho0) + 0.5*alpha*C*ny*nz - m14);
			m15 = m15 + rlx_setA*( (jx*jz/rho0) + 0.5*alpha*C*nx*nz - m15);
			m16 = m16 + rlx_setB*( - m16);
			m17 = m17 + rlx_setB*( - m17);
			m18 = m18 + rlx_setB*( - m18);
			//.................inverse transformation......................................................

			// q=0
			fq = mrt_V1*rho-mrt_V2*m1+mrt_V3*m2;
			dist[n] = fq;

			// q = 1
			fq = mrt_V1*rho-mrt_V4*m1-mrt_V5*m2+0.1*(jx-m4)+mrt_V6*(m9-m10)+0.16666666*Fx;
			//nread = neighborList[n+Np];
			dist[nr2] = fq;

			// q=2
			fq = mrt_V1*rho-mrt_V4*m1-mrt_V5*m2+0.1*(m4-jx)+mrt_V6*(m9-m10) -  0.16666666*Fx;
			//nread = neighborList[n];
			dist[nr1] = fq;

			// q = 3
			fq = mrt_V1*rho-mrt_V4*m1-mrt_V5*m2+0.1*(jy-m6)+mrt_V7*(m10-m9)+mrt_V8*(m11-m12) + 0.16666666*Fy;
			//nread = neighborList[n+3*Np];
			dist[nr4] = fq;

			// q = 4
			fq = mrt_V1*rho-mrt_V4*m1-mrt_V5*m2+0.1*(m6-jy)+mrt_V7*(m10-m9)+mrt_V8*(m11-m12) - 0.16666666*Fy;
			//nread = neighborList[n+2*Np];
			dist[nr3] = fq;

			// q = 5
			fq = mrt_V1*rho-mrt_V4*m1-mrt_V5*m2+0.1*(jz-m8)+mrt_V7*(m10-m9)+mrt_V8*(m12-m11) + 0.16666666*Fz;
			//nread = neighborList[n+5*Np];
			dist[nr6] = fq;

			// q = 6
			fq = mrt_V1*rho-mrt_V4*m1-mrt_V5*m2+0.1*(m8-jz)+mrt_V7*(m10-m9)+mrt_V8*(m12-m11) - 0.16666666*Fz;
			//nread = neighborList[n+4*Np];
			dist[nr5] = fq;

			// q = 7
			fq = mrt_V1*rho+mrt_V9*m1+mrt_V10*m2+0.1*(jx+jy)+0.025*(m4+m6)+
					mrt_V7*m9+mrt_V11*m10+mrt_V8*m11+mrt_V12*m12+0.25*m13+0.125*(m16-m17) + 0.08333333333*(Fx+Fy);
			//nread = neighborList[n+7*Np];
			dist[nr8] = fq;

			// q = 8
			fq = mrt_V1*rho+mrt_V9*m1+mrt_V10*m2-0.1*(jx+jy)-0.025*(m4+m6) +mrt_V7*m9+mrt_V11*m10+mrt_V8*m11
					+mrt_V12*m12+0.25*m13+0.125*(m17-m16) - 0.08333333333*(Fx+Fy);
			//nread = neighborList[n+6*Np];
			dist[nr7] = fq;

			// q = 9
			fq = mrt_V1*rho+mrt_V9*m1+mrt_V10*m2+0.1*(jx-jy)+0.025*(m4-m6)+
					mrt_V7*m9+mrt_V11*m10+mrt_V8*m11+mrt_V12*m12-0.25*m13+0.125*(m16+m17) + 0.08333333333*(Fx-Fy);
			//nread = neighborList[n+9*Np];
			dist[nr10] = fq;

			// q = 10
			fq = mrt_V1*rho+mrt_V9*m1+mrt_V10*m2+0.1*(jy-jx)+0.025*(m6-m4)+
					mrt_V7*m9+mrt_V11*m10+mrt_V8*m11+mrt_V12*m12-0.25*m13-0.125*(m16+m17)- 0.08333333333*(Fx-Fy);
			//nread = neighborList[n+8*Np];
			dist[nr9] = fq;

			// q = 11
			fq = mrt_V1*rho+mrt_V9*m1
					+mrt_V10*m2+0.1*(jx+jz)+0.025*(m4+m8)
					+mrt_V7*m9+mrt_V11*m10-mrt_V8*m11
					-mrt_V12*m12+0.25*m15+0.125*(m18-m16) + 0.08333333333*(Fx+Fz);
			//nread = neighborList[n+11*Np];
			dist[nr12] = fq;

			// q = 12
			fq = mrt_V1*rho+mrt_V9*m1+mrt_V10*m2-0.1*(jx+jz)-0.025*(m4+m8)+
					mrt_V7*m9+mrt_V11*m10-mrt_V8*m11-mrt_V12*m12+0.25*m15+0.125*(m16-m18) - 0.08333333333*(Fx+Fz);
			//nread = neighborList[n+10*Np];
			dist[nr11]= fq;

			// q = 13
			fq = mrt_V1*rho+mrt_V9*m1
					+mrt_V10*m2+0.1*(jx-jz)+0.025*(m4-m8)
					+mrt_V7*m9+mrt_V11*m10-mrt_V8*m11
					-mrt_V12*m12-0.25*m15-0.125*(m16+m18) + 0.08333333333*(Fx-Fz);
			//nread = neighborList[n+13*Np];
			dist[nr14] = fq;

			// q= 14
			fq = mrt_V1*rho+mrt_V9*m1
					+mrt_V10*m2+0.1*(jz-jx)+0.025*(m8-m4)
					+mrt_V7*m9+mrt_V11*m10-mrt_V8*m11
					-mrt_V12*m12-0.25*m15+0.125*(m16+m18) - 0.08333333333*(Fx-Fz);
			//nread = neighborList[n+12*Np];
			dist[nr13] = fq;


			// q = 15
			fq = mrt_V1*rho+mrt_V9*m1
					+mrt_V10*m2+0.1*(jy+jz)+0.025*(m6+m8)
					-mrt_V6*m9-mrt_V7*m10+0.25*m14+0.125*(m17-m18) + 0.08333333333*(Fy+Fz);
			nread = neighborList[n+15*Np];
			dist[nread] = fq;

			// q = 16
			fq =  mrt_V1*rho+mrt_V9*m1
					+mrt_V10*m2-0.1*(jy+jz)-0.025*(m6+m8)
					-mrt_V6*m9-mrt_V7*m10+0.25*m14+0.125*(m18-m17)- 0.08333333333*(Fy+Fz);
			nread = neighborList[n+14*Np];
			dist[nread] = fq;


			// q = 17
			fq = mrt_V1*rho+mrt_V9*m1
					+mrt_V10*m2+0.1*(jy-jz)+0.025*(m6-m8)
					-mrt_V6*m9-mrt_V7*m10-0.25*m14+0.125*(m17+m18) + 0.08333333333*(Fy-Fz);
			nread = neighborList[n+17*Np];
			dist[nread] = fq;

			// q = 18
			fq = mrt_V1*rho+mrt_V9*m1
					+mrt_V10*m2+0.1*(jz-jy)+0.025*(m8-m6)
					-mrt_V6*m9-mrt_V7*m10-0.25*m14-0.125*(m17+m18) - 0.08333333333*(Fy-Fz);
			nread = neighborList[n+16*Np];
			dist[nread] = fq;

			// write the velocity 
			ux = jx / rho0;
			uy = jy / rho0;
			uz = jz / rho0;
			Velocity[n] = ux;
			Velocity[Np+n] = uy;
			Velocity[2*Np+n] = uz;

			// Instantiate mass transport distributions
			// Stationary value - distribution 0
			nAB = 1.0/(nA+nB);
			Aq[n] = 0.3333333333333333*nA;
			Bq[n] = 0.3333333333333333*nB;

			//...............................................
			// q = 0,2,4
			// Cq = {1,0,0}, {0,1,0}, {0,0,1}
			delta = beta*nA*nB*nAB*0.1111111111111111*nx;
			if (!(nA*nB*nAB>0)) delta=0;
			a1 = nA*(0.1111111111111111*(1+4.5*ux))+delta;
			b1 = nB*(0.1111111111111111*(1+4.5*ux))-delta;
			a2 = nA*(0.1111111111111111*(1-4.5*ux))-delta;
			b2 = nB*(0.1111111111111111*(1-4.5*ux))+delta;

			// q = 1
			//nread = neighborList[n+Np];
			Aq[nr2] = a1;
			Bq[nr2] = b1;
			// q=2
			//nread = neighborList[n];
			Aq[nr1] = a2;
			Bq[nr1] = b2;

			//...............................................
			// Cq = {0,1,0}
			delta = beta*nA*nB*nAB*0.1111111111111111*ny;
			if (!(nA*nB*nAB>0)) delta=0;
			a1 = nA*(0.1111111111111111*(1+4.5*uy))+delta;
			b1 = nB*(0.1111111111111111*(1+4.5*uy))-delta;
			a2 = nA*(0.1111111111111111*(1-4.5*uy))-delta;
			b2 = nB*(0.1111111111111111*(1-4.5*uy))+delta;

			// q = 3
			//nread = neighborList[n+3*Np];
			Aq[nr4] = a1;
			Bq[nr4] = b1;
			// q = 4
			//nread = neighborList[n+2*Np];
			Aq[nr3] = a2;
			Bq[nr3] = b2;

			//...............................................
			// q = 4
			// Cq = {0,0,1}
			delta = beta*nA*nB*nAB*0.1111111111111111*nz;
			if (!(nA*nB*nAB>0)) delta=0;
			a1 = nA*(0.1111111111111111*(1+4.5*uz))+delta;
			b1 = nB*(0.1111111111111111*(1+4.5*uz))-delta;
			a2 = nA*(0.1111111111111111*(1-4.5*uz))-delta;
			b2 = nB*(0.1111111111111111*(1-4.5*uz))+delta;

			// q = 5
			//nread = neighborList[n+5*Np];
			Aq[nr6] = a1;
			Bq[nr6] = b1;
			// q = 6
			//nread = neighborList[n+4*Np];
			Aq[nr5] = a2;
			Bq[nr5] = b2;
			//...............................................
		}
	}
}

__global__  void dvc_ScaLBL_D3Q7_AAodd_PhaseField(int *neighborList, int *Map, double *Aq, double *Bq, 
		double *Den, double *Phi, int start, int finish, int Np){
	int idx,n,nread;
	double fq,nA,nB;

	int S = Np/NBLOCKS/NTHREADS + 1;
	for (int s=0; s<S; s++){
		//........Get 1-D index for this thread....................
		n =  S*blockIdx.x*blockDim.x + s*blockDim.x + threadIdx.x + start;
		if (n<finish) {
			//..........Compute the number density for each component ............
			// q=0
			fq = Aq[n];
			nA = fq;
			fq = Bq[n];
			nB = fq;
			
			// q=1
			nread = neighborList[n]; 
			fq = Aq[nread];
			nA += fq;
			fq = Bq[nread]; 
			nB += fq;
			
			// q=2
			nread = neighborList[n+Np]; 
			fq = Aq[nread];  
			nA += fq;
			fq = Bq[nread]; 
			nB += fq;
			
			// q=3
			nread = neighborList[n+2*Np]; 
			fq = Aq[nread];
			nA += fq;
			fq = Bq[nread];
			nB += fq;
			
			// q = 4
			nread = neighborList[n+3*Np]; 
			fq = Aq[nread];
			nA += fq;
			fq = Bq[nread];
			nB += fq;

			// q=5
			nread = neighborList[n+4*Np];
			fq = Aq[nread];
			nA += fq;
			fq = Bq[nread];
			nB += fq;
			
			// q = 6
			nread = neighborList[n+5*Np];
			fq = Aq[nread];
			nA += fq;
			fq = Bq[nread];
			nB += fq;

			// save the number densities
			Den[n] = nA;
			Den[Np+n] = nB;

			// save the phase indicator field
			idx = Map[n];
			Phi[idx] = (nA-nB)/(nA+nB); 
		}
	}
}

__global__  void dvc_ScaLBL_D3Q7_AAeven_PhaseField(int *Map, double *Aq, double *Bq, double *Den, double *Phi, 
		int start, int finish, int Np){
	int idx,n;
	double fq,nA,nB;

	int S = Np/NBLOCKS/NTHREADS + 1;
	for (int s=0; s<S; s++){
		//........Get 1-D index for this thread....................
		n =  S*blockIdx.x*blockDim.x + s*blockDim.x + threadIdx.x + start;
		if (n<finish) {
			// compute number density for each component
			// q=0
			fq = Aq[n];
			nA = fq;
			fq = Bq[n];
			nB = fq;
			
			// q=1
			fq = Aq[2*Np+n];
			nA += fq;
			fq = Bq[2*Np+n];
			nB += fq;

			// q=2
			fq = Aq[1*Np+n];
			nA += fq;
			fq = Bq[1*Np+n];
			nB += fq;

			// q=3
			fq = Aq[4*Np+n];
			nA += fq;
			fq = Bq[4*Np+n];
			nB += fq;

			// q = 4
			fq = Aq[3*Np+n];
			nA += fq;
			fq = Bq[3*Np+n];
			nB += fq;
			
			// q=5
			fq = Aq[6*Np+n];
			nA += fq;
			fq = Bq[6*Np+n];
			nB += fq;
			
			// q = 6
			fq = Aq[5*Np+n];
			nA += fq;
			fq = Bq[5*Np+n];
			nB += fq;

			// save the number densities
			Den[n] = nA;
			Den[Np+n] = nB;

			// save the phase indicator field
			idx = Map[n];
			Phi[idx] = (nA-nB)/(nA+nB); 	
		}
	}
}

__global__ void dvc_ScaLBL_PhaseField_Init(int *Map, double *Phi, double *Den, double *Aq, double *Bq, int start, int finish, int Np){
	int idx,n;
	double phi,nA,nB;

	int S = Np/NBLOCKS/NTHREADS + 1;
	for (int s=0; s<S; s++){
		//........Get 1-D index for this thread....................
		idx =  S*blockIdx.x*blockDim.x + s*blockDim.x + threadIdx.x + start;
		if (idx<finish) {

			n = Map[idx];
			phi = Phi[n];
            if (phi > 1.f){
                    nA = 1.0; nB = 0.f;
            }
            else if (phi < -1.f){
                    nB = 1.0; nA = 0.f;
            }
            else{
                    nA=0.5*(phi+1.f);
                    nB=0.5*(1.f-phi);
            }
			Den[idx] = nA;
			Den[Np+idx] = nB;

			Aq[idx]=0.3333333333333333*nA;
			Aq[Np+idx]=0.1111111111111111*nA;
			Aq[2*Np+idx]=0.1111111111111111*nA;
			Aq[3*Np+idx]=0.1111111111111111*nA;
			Aq[4*Np+idx]=0.1111111111111111*nA;
			Aq[5*Np+idx]=0.1111111111111111*nA;
			Aq[6*Np+idx]=0.1111111111111111*nA;

			Bq[idx]=0.3333333333333333*nB;
			Bq[Np+idx]=0.1111111111111111*nB;
			Bq[2*Np+idx]=0.1111111111111111*nB;
			Bq[3*Np+idx]=0.1111111111111111*nB;
			Bq[4*Np+idx]=0.1111111111111111*nB;
			Bq[5*Np+idx]=0.1111111111111111*nB;
			Bq[6*Np+idx]=0.1111111111111111*nB;
		}
	}
}

extern "C" void ScaLBL_SetSlice_z(double *Phi, double value, int Nx, int Ny, int Nz, int Slice){
	int GRID = Nx*Ny / 512 + 1;
	dvc_ScaLBL_SetSlice_z<<<GRID,512>>>(Phi,value,Nx,Ny,Nz,Slice);
}
extern "C" void ScaLBL_CopySlice_z(double *Phi, int Nx, int Ny, int Nz, int Source, int Dest){
	int GRID = Nx*Ny / 512 + 1;
	dvc_ScaLBL_CopySlice_z<<<GRID,512>>>(Phi,Nx,Ny,Nz,Source,Dest);
}
extern "C" void ScaLBL_Color_BC(int *list, int *Map, double *Phi, double *Den, double vA, double vB, int count, int Np){
    int GRID = count / 512 + 1;
    dvc_ScaLBL_Color_BC<<<GRID,512>>>(list, Map, Phi, Den, vA, vB, count, Np);
	hipError_t err = hipGetLastError();
	if (hipSuccess != err){
		printf("CUDA error in ScaLBL_Color_BC: %s \n",hipGetErrorString(err));
	}
}
// Pressure Boundary Conditions Functions

extern "C" void ScaLBL_D3Q19_AAeven_Color(int *Map, double *dist, double *Aq, double *Bq, double *Den, double *Phi,
		double *Vel, double rhoA, double rhoB, double tauA, double tauB, double alpha, double beta,
		double Fx, double Fy, double Fz, int strideY, int strideZ, int start, int finish, int Np){

	hipProfilerStart();
	hipFuncSetCacheConfig(reinterpret_cast<const void*>(dvc_ScaLBL_D3Q19_AAeven_Color), hipFuncCachePreferL1);

	dvc_ScaLBL_D3Q19_AAeven_Color<<<NBLOCKS,NTHREADS >>>(Map, dist, Aq, Bq, Den, Phi, Vel, rhoA, rhoB, tauA, tauB, 
			alpha, beta, Fx, Fy, Fz, strideY, strideZ, start, finish, Np);
	hipError_t err = hipGetLastError();
	if (hipSuccess != err){
		printf("CUDA error in ScaLBL_D3Q19_AAeven_Color: %s \n",hipGetErrorString(err));
	}
	hipProfilerStop();

}

extern "C" void ScaLBL_D3Q19_AAodd_Color(int *d_neighborList, int *Map, double *dist, double *Aq, double *Bq, double *Den, 
		double *Phi, double *Vel, double rhoA, double rhoB, double tauA, double tauB, double alpha, double beta,
		double Fx, double Fy, double Fz, int strideY, int strideZ, int start, int finish, int Np){

	hipProfilerStart();
	hipFuncSetCacheConfig(reinterpret_cast<const void*>(dvc_ScaLBL_D3Q19_AAodd_Color), hipFuncCachePreferL1);
	
	dvc_ScaLBL_D3Q19_AAodd_Color<<<NBLOCKS,NTHREADS >>>(d_neighborList, Map, dist, Aq, Bq, Den, Phi, Vel, 
			rhoA, rhoB, tauA, tauB, alpha, beta, Fx, Fy, Fz, strideY, strideZ, start, finish, Np);

	hipError_t err = hipGetLastError();
	if (hipSuccess != err){
		printf("CUDA error in ScaLBL_D3Q19_AAodd_Color: %s \n",hipGetErrorString(err));
	}
	hipProfilerStop();
}

extern "C" void ScaLBL_D3Q7_AAodd_PhaseField(int *NeighborList, int *Map, double *Aq, double *Bq, 
		double *Den, double *Phi, int start, int finish, int Np){

	hipProfilerStart();
	dvc_ScaLBL_D3Q7_AAodd_PhaseField<<<NBLOCKS,NTHREADS >>>(NeighborList, Map, Aq, Bq, Den, Phi, start, finish, Np);

	hipError_t err = hipGetLastError();
	if (hipSuccess != err){
		printf("CUDA error in ScaLBL_D3Q7_AAodd_PhaseField: %s \n",hipGetErrorString(err));
	}
	hipProfilerStop();
}

extern "C" void ScaLBL_D3Q7_AAeven_PhaseField(int *Map, double *Aq, double *Bq, double *Den, double *Phi, 
		int start, int finish, int Np){

	hipProfilerStart();
	dvc_ScaLBL_D3Q7_AAeven_PhaseField<<<NBLOCKS,NTHREADS >>>(Map, Aq, Bq, Den, Phi, start, finish, Np);
	hipError_t err = hipGetLastError();
	if (hipSuccess != err){
		printf("CUDA error in ScaLBL_D3Q7_AAeven_PhaseField: %s \n",hipGetErrorString(err));
	}
	hipProfilerStop();

}

extern "C" void ScaLBL_PhaseField_Init(int *Map, double *Phi, double *Den, double *Aq, double *Bq, int start, int finish, int Np){
	dvc_ScaLBL_PhaseField_Init<<<NBLOCKS,NTHREADS >>>(Map, Phi, Den, Aq, Bq, start, finish, Np); 
	hipError_t err = hipGetLastError();
	if (hipSuccess != err){
		printf("CUDA error in ScaLBL_PhaseField_Init: %s \n",hipGetErrorString(err));
	}
}


