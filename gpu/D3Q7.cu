
#include <hip/hip_runtime.h>
/*
  Copyright 2013--2018 James E. McClure, Virginia Polytechnic & State University

  This file is part of the Open Porous Media project (OPM).
  OPM is free software: you can redistribute it and/or modify
  it under the terms of the GNU General Public License as published by
  the Free Software Foundation, either version 3 of the License, or
  (at your option) any later version.
  OPM is distributed in the hope that it will be useful,
  but WITHOUT ANY WARRANTY; without even the implied warranty of
  MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the
  GNU General Public License for more details.
  You should have received a copy of the GNU General Public License
  along with OPM.  If not, see <http://www.gnu.org/licenses/>.
*/
// GPU Functions for D3Q7 Lattice Boltzmann Methods

#define NBLOCKS 8192
#define NTHREADS 256

__global__  void dvc_ScaLBL_Scalar_Pack(int *list, int count, double *sendbuf, double *Data, int N){
	//....................................................................................
	// Pack distribution q into the send buffer for the listed lattice sites
	// dist may be even or odd distributions stored by stream layout
	//....................................................................................
	int idx,n;
	idx = blockIdx.x*blockDim.x + threadIdx.x;
	if (idx<count){
		n = list[idx];
		sendbuf[idx] = Data[n];
	}
}
__global__  void dvc_ScaLBL_Scalar_Unpack(int *list, int count, double *recvbuf, double *Data, int N){
	//....................................................................................
	// Pack distribution q into the send buffer for the listed lattice sites
	// dist may be even or odd distributions stored by stream layout
	//....................................................................................
	int idx,n;
	idx = blockIdx.x*blockDim.x + threadIdx.x;
	if (idx<count){
		n = list[idx];
		Data[n] = recvbuf[idx];
	}
}

__global__  void dvc_ScaLBL_PackDenD3Q7(int *list, int count, double *sendbuf, int number, double *Data, int N){
	//....................................................................................
	// Pack distribution into the send buffer for the listed lattice sites
	//....................................................................................
	int idx,n,component;
	idx = blockIdx.x*blockDim.x + threadIdx.x;
	if (idx<count){
		for (component=0; component<number; component++){
			n = list[idx];
			sendbuf[idx*number+component] = Data[number*n+component];
			Data[number*n+component] = 0.0;	// Set the data value to zero once it's in the buffer!
		}
	}
}


__global__ void dvc_ScaLBL_UnpackDenD3Q7(int *list, int count, double *recvbuf, int number, double *Data, int N){
	//....................................................................................
	// Unack distribution from the recv buffer
	// Sum to the existing density value
	//....................................................................................
	int idx,n,component;
	idx = blockIdx.x*blockDim.x + threadIdx.x;
	if (idx<count){
			for (component=0; component<number; component++){
			n = list[idx];
			Data[number*n+component] += recvbuf[idx*number+component];
		}
	}
}

__global__ void dvc_ScaLBL_D3Q7_Unpack(int q,  int *list,  int start, int count,
		double *recvbuf, double *dist, int N){
	//....................................................................................
	// Unpack distribution from the recv buffer
	// Distribution q matche Cqx, Cqy, Cqz
	// swap rule means that the distributions in recvbuf are OPPOSITE of q
	// dist may be even or odd distributions stored by stream layout
	//....................................................................................
	int n,idx;
	idx = blockIdx.x*blockDim.x + threadIdx.x;
	if (idx<count){
		// Get the value from the list -- note that n is the index is from the send (non-local) process
		n = list[idx];
		// unpack the distribution to the proper location
		if (!(n<0)) { dist[q*N+n] = recvbuf[start+idx];
		//printf("%f \n",,dist[q*N+n]);
		}
	}
}


extern "C" void ScaLBL_D3Q7_Unpack(int q, int *list,  int start, int count, double *recvbuf, double *dist, int N){
	int GRID = count / 512 + 1;
	dvc_ScaLBL_D3Q7_Unpack <<<GRID,512 >>>(q, list, start, count, recvbuf, dist, N);
}

extern "C" void ScaLBL_Scalar_Pack(int *list, int count, double *sendbuf, double *Data, int N){
	int GRID = count / 512 + 1;
	dvc_ScaLBL_Scalar_Pack <<<GRID,512 >>>(list, count, sendbuf, Data, N);
}

extern "C" void ScaLBL_Scalar_Unpack(int *list, int count, double *recvbuf, double *Data, int N){
	int GRID = count / 512 + 1;
	dvc_ScaLBL_Scalar_Unpack <<<GRID,512 >>>(list, count, recvbuf, Data, N);
}
extern "C" void ScaLBL_PackDenD3Q7(int *list, int count, double *sendbuf, int number, double *Data, int N){
	int GRID = count / 512 + 1;
	dvc_ScaLBL_PackDenD3Q7 <<<GRID,512 >>>(list, count, sendbuf, number, Data, N);
}

extern "C" void ScaLBL_UnpackDenD3Q7(int *list, int count, double *recvbuf, int number, double *Data, int N){
	int GRID = count / 512 + 1;
	dvc_ScaLBL_UnpackDenD3Q7 <<<GRID,512 >>>(list, count, recvbuf, number, Data, N);
}

