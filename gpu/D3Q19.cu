#include "hip/hip_runtime.h"
/*
  Copyright 2013--2018 James E. McClure, Virginia Polytechnic & State University

  This file is part of the Open Porous Media project (OPM).
  OPM is free software: you can redistribute it and/or modify
  it under the terms of the GNU General Public License as published by
  the Free Software Foundation, either version 3 of the License, or
  (at your option) any later version.
  OPM is distributed in the hope that it will be useful,
  but WITHOUT ANY WARRANTY; without even the implied warranty of
  MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the
  GNU General Public License for more details.
  You should have received a copy of the GNU General Public License
  along with OPM.  If not, see <http://www.gnu.org/licenses/>.
*/
#include <stdio.h>
#include <hip/hip_cooperative_groups.h>

#define NBLOCKS 8192
#define NTHREADS 256

/*
1. constants that are known at compile time should be defined using preprocessor macros (e.g. #define) or via C/C++ const variables at global/file scope.
2. Usage of __constant__ memory may be beneficial for programs who use certain values that don't change for the duration of the kernel and for which certain access patterns are present (e.g. all threads access the same value at the same time). This is not better or faster than constants that satisfy the requirements of item 1 above.
3. If the number of choices to be made by a program are relatively small in number, and these choices affect kernel execution, one possible approach for additional compile-time optimization would be to use templated code/kernels
 */

__constant__ __device__ double mrt_V1=0.05263157894736842;
__constant__ __device__ double mrt_V2=0.012531328320802;
__constant__ __device__ double mrt_V3=0.04761904761904762;
__constant__ __device__ double mrt_V4=0.004594820384294068;
__constant__ __device__ double mrt_V5=0.01587301587301587;
__constant__ __device__ double mrt_V6=0.0555555555555555555555555;
__constant__ __device__ double mrt_V7=0.02777777777777778;
__constant__ __device__ double mrt_V8=0.08333333333333333;
__constant__ __device__ double mrt_V9=0.003341687552213868;
__constant__ __device__ double mrt_V10=0.003968253968253968;
__constant__ __device__ double mrt_V11=0.01388888888888889;
__constant__ __device__ double mrt_V12=0.04166666666666666;


// functionality for parallel reduction in Flux BC routines -- probably should be re-factored to another location
// functions copied from https://devblogs.nvidia.com/parallelforall/faster-parallel-reductions-kepler/

//__shared__ double Transform[722]=
//	   {};

#if !defined(__CUDA_ARCH__) || __CUDA_ARCH__ >= 600
#else
__device__ double atomicAdd(double* address, double val) { 
   unsigned long long int* address_as_ull = (unsigned long long int*)address;
   unsigned long long int old = *address_as_ull, assumed;

   do {
      assumed = old;
      old = atomicCAS(address_as_ull, assumed, __double_as_longlong(val+__longlong_as_double(assumed)));
   } while (assumed != old);
   return __longlong_as_double(old);
}
#endif

using namespace cooperative_groups;
__device__ double reduce_sum(thread_group g, double *temp, double val)
{
    int lane = g.thread_rank();

    // Each iteration halves the number of active threads
    // Each thread adds its partial sum[i] to sum[lane+i]
    for (int i = g.size() / 2; i > 0; i /= 2)
    {
        temp[lane] = val;
        g.sync(); // wait for all threads to store
        if(lane<i) val += temp[lane + i];
        g.sync(); // wait for all threads to load
    }
    return val; // note: only thread 0 will return full sum
}

__device__ double thread_sum(double *input, double n) 
{
    double sum = 0;

    for(int i = blockIdx.x * blockDim.x + threadIdx.x;
        i < n / 4; 
        i += blockDim.x * gridDim.x)
    {
        int4 in = ((int4*)input)[i];
        sum += in.x + in.y + in.z + in.w;
    }
    return sum;
}

__global__ void sum_kernel_block(double *sum, double *input, int n)
{
	double my_sum = thread_sum(input, n);

    extern __shared__ double temp[];
    thread_group g = this_thread_block();
    double block_sum = reduce_sum(g, temp, my_sum);

    if (g.thread_rank() == 0) atomicAdd(sum, block_sum);
}

__inline__ __device__
double warpReduceSum(double val) {
	for (int offset = warpSize/2; offset > 0; offset /= 2)
		val += __shfl_down_sync(0xFFFFFFFF, val, offset, 32);
	return val;
}

__inline__ __device__
double blockReduceSum(double val) {

	static __shared__ double shared[32]; // Shared mem for 32 partial sums
	int lane = threadIdx.x % warpSize;
	int wid = threadIdx.x / warpSize;

	val = warpReduceSum(val);     // Each warp performs partial reduction

	if (lane==0) shared[wid]=val; // Write reduced value to shared memory

	__syncthreads();              // Wait for all partial reductions

	//read from shared memory only if that warp existed
	val = (threadIdx.x < blockDim.x / warpSize) ? shared[lane] : 0;

	if (wid==0) val = warpReduceSum(val); //Final reduce within first warp

	return val;
}

__global__ void deviceReduceKernel(double *in, double* out, int N) {
	double sum = 0;
	//reduce multiple elements per thread
	for (int i = blockIdx.x * blockDim.x + threadIdx.x;
			i < N;
			i += blockDim.x * gridDim.x) {
		sum += in[i];
	}
	sum = blockReduceSum(sum);
	if (threadIdx.x==0)
		out[blockIdx.x]=sum;
}

__global__ void dvc_ScaLBL_D3Q19_Pack(int q, int *list, int start, int count, double *sendbuf, double *dist, int N){
	//....................................................................................
	// Pack distribution q into the send buffer for the listed lattice sites
	// dist may be even or odd distributions stored by stream layout
	//....................................................................................
	int idx,n;
	idx = blockIdx.x*blockDim.x + threadIdx.x;
	if (idx<count){
		n = list[idx];
		sendbuf[start+idx] = dist[q*N+n];
		//printf("%f \n",dist[q*N+n]);
	}

}

__global__ void dvc_ScaLBL_D3Q19_Unpack(int q,  int *list,  int start, int count,
		double *recvbuf, double *dist, int N){
	//....................................................................................
	// Unpack distribution from the recv buffer
	// Distribution q matche Cqx, Cqy, Cqz
	// swap rule means that the distributions in recvbuf are OPPOSITE of q
	// dist may be even or odd distributions stored by stream layout
	//....................................................................................
	int n,idx;
	idx = blockIdx.x*blockDim.x + threadIdx.x;
	if (idx<count){
		// Get the value from the list -- note that n is the index is from the send (non-local) process
		n = list[start+idx];
		// unpack the distribution to the proper location
		if (!(n<0)) { dist[q*N+n] = recvbuf[start+idx];
		//printf("%f \n",,dist[q*N+n]);
		}
	}
}

__global__ void dvc_ScaLBL_D3Q19_Init(double *dist, int Np)
{
	int n;
	int S = Np/NBLOCKS/NTHREADS + 1;
	for (int s=0; s<S; s++){
		//........Get 1-D index for this thread....................
		n = S*blockIdx.x*blockDim.x + s*blockDim.x + threadIdx.x;
		if (n<Np ){
			dist[n] = 0.3333333333333333;
			dist[Np+n] = 0.055555555555555555;		//double(100*n)+1.f;
			dist[2*Np+n] = 0.055555555555555555;	//double(100*n)+2.f;
			dist[3*Np+n] = 0.055555555555555555;	//double(100*n)+3.f;
			dist[4*Np+n] = 0.055555555555555555;	//double(100*n)+4.f;
			dist[5*Np+n] = 0.055555555555555555;	//double(100*n)+5.f;
			dist[6*Np+n] = 0.055555555555555555;	//double(100*n)+6.f;
			dist[7*Np+n] = 0.0277777777777778;   //double(100*n)+7.f;
			dist[8*Np+n] = 0.0277777777777778;   //double(100*n)+8.f;
			dist[9*Np+n] = 0.0277777777777778;   //double(100*n)+9.f;
			dist[10*Np+n] = 0.0277777777777778;  //double(100*n)+10.f;
			dist[11*Np+n] = 0.0277777777777778;  //double(100*n)+11.f;
			dist[12*Np+n] = 0.0277777777777778;  //double(100*n)+12.f;
			dist[13*Np+n] = 0.0277777777777778;  //double(100*n)+13.f;
			dist[14*Np+n] = 0.0277777777777778;  //double(100*n)+14.f;
			dist[15*Np+n] = 0.0277777777777778;  //double(100*n)+15.f;
			dist[16*Np+n] = 0.0277777777777778;  //double(100*n)+16.f;
			dist[17*Np+n] = 0.0277777777777778;  //double(100*n)+17.f;
			dist[18*Np+n] = 0.0277777777777778;  //double(100*n)+18.f;
		}
	}
}



__global__ void 
dvc_ScaLBL_AAodd_MRT(int *neighborList, double *dist, int start, int finish, int Np, double rlx_setA, double rlx_setB, double Fx, double Fy, double Fz) {

	int n;
	double fq;
	// conserved momemnts
	double rho,jx,jy,jz;
	// non-conserved moments
	double m1,m2,m4,m6,m8,m9,m10,m11,m12,m13,m14,m15,m16,m17,m18;

	int nread;
	int S = Np/NBLOCKS/NTHREADS+1;

	for (int s=0; s<S; s++){
		//........Get 1-D index for this thread....................
		n =  S*blockIdx.x*blockDim.x + s*blockDim.x + threadIdx.x + start;
		if (n<finish) {
			// q=0
			fq = dist[n];
			rho = fq;
			m1  = -30.0*fq;
			m2  = 12.0*fq;

			// q=1
			nread = neighborList[n]; // neighbor 2 ( > 10Np => odd part of dist)
			fq = dist[nread]; // reading the f1 data into register fq
			//fp = dist[10*Np+n];
			rho += fq;
			m1 -= 11.0*fq;
			m2 -= 4.0*fq;
			jx = fq;
			m4 = -4.0*fq;
			m9 = 2.0*fq;
			m10 = -4.0*fq;

			// f2 = dist[10*Np+n];
			nread = neighborList[n+Np]; // neighbor 1 ( < 10Np => even part of dist)
			fq = dist[nread];  // reading the f2 data into register fq
			//fq = dist[Np+n];
			rho += fq;
			m1 -= 11.0*(fq);
			m2 -= 4.0*(fq);
			jx -= fq;
			m4 += 4.0*(fq);
			m9 += 2.0*(fq);
			m10 -= 4.0*(fq);

			// q=3
			nread = neighborList[n+2*Np]; // neighbor 4
			fq = dist[nread];
			//fq = dist[11*Np+n];
			rho += fq;
			m1 -= 11.0*fq;
			m2 -= 4.0*fq;
			jy = fq;
			m6 = -4.0*fq;
			m9 -= fq;
			m10 += 2.0*fq;
			m11 = fq;
			m12 = -2.0*fq;

			// q = 4
			nread = neighborList[n+3*Np]; // neighbor 3
			fq = dist[nread];
			//fq = dist[2*Np+n];
			rho+= fq;
			m1 -= 11.0*fq;
			m2 -= 4.0*fq;
			jy -= fq;
			m6 += 4.0*fq;
			m9 -= fq;
			m10 += 2.0*fq;
			m11 += fq;
			m12 -= 2.0*fq;

			// q=5
			nread = neighborList[n+4*Np];
			fq = dist[nread];
			//fq = dist[12*Np+n];
			rho += fq;
			m1 -= 11.0*fq;
			m2 -= 4.0*fq;
			jz = fq;
			m8 = -4.0*fq;
			m9 -= fq;
			m10 += 2.0*fq;
			m11 -= fq;
			m12 += 2.0*fq;


			// q = 6
			nread = neighborList[n+5*Np];
			fq = dist[nread];
			//fq = dist[3*Np+n];
			rho+= fq;
			m1 -= 11.0*fq;
			m2 -= 4.0*fq;
			jz -= fq;
			m8 += 4.0*fq;
			m9 -= fq;
			m10 += 2.0*fq;
			m11 -= fq;
			m12 += 2.0*fq;

			// q=7
			nread = neighborList[n+6*Np];
			fq = dist[nread];
			//fq = dist[13*Np+n];
			rho += fq;
			m1 += 8.0*fq;
			m2 += fq;
			jx += fq;
			m4 += fq;
			jy += fq;
			m6 += fq;
			m9  += fq;
			m10 += fq;
			m11 += fq;
			m12 += fq;
			m13 = fq;
			m16 = fq;
			m17 = -fq;

			// q = 8
			nread = neighborList[n+7*Np];
			fq = dist[nread];
			//fq = dist[4*Np+n];
			rho += fq;
			m1 += 8.0*fq;
			m2 += fq;
			jx -= fq;
			m4 -= fq;
			jy -= fq;
			m6 -= fq;
			m9 += fq;
			m10 += fq;
			m11 += fq;
			m12 += fq;
			m13 += fq;
			m16 -= fq;
			m17 += fq;

			// q=9
			nread = neighborList[n+8*Np];
			fq = dist[nread];
			//fq = dist[14*Np+n];
			rho += fq;
			m1 += 8.0*fq;
			m2 += fq;
			jx += fq;
			m4 += fq;
			jy -= fq;
			m6 -= fq;
			m9 += fq;
			m10 += fq;
			m11 += fq;
			m12 += fq;
			m13 -= fq;
			m16 += fq;
			m17 += fq;

			// q = 10
			nread = neighborList[n+9*Np];
			fq = dist[nread];
			//fq = dist[5*Np+n];
			rho += fq;
			m1 += 8.0*fq;
			m2 += fq;
			jx -= fq;
			m4 -= fq;
			jy += fq;
			m6 += fq;
			m9 += fq;
			m10 += fq;
			m11 += fq;
			m12 += fq;
			m13 -= fq;
			m16 -= fq;
			m17 -= fq;

			// q=11
			nread = neighborList[n+10*Np];
			fq = dist[nread];
			//fq = dist[15*Np+n];
			rho += fq;
			m1 += 8.0*fq;
			m2 += fq;
			jx += fq;
			m4 += fq;
			jz += fq;
			m8 += fq;
			m9 += fq;
			m10 += fq;
			m11 -= fq;
			m12 -= fq;
			m15 = fq;
			m16 -= fq;
			m18 = fq;

			// q=12
			nread = neighborList[n+11*Np];
			fq = dist[nread];
			//fq = dist[6*Np+n];
			rho += fq;
			m1 += 8.0*fq;
			m2 += fq;
			jx -= fq;
			m4 -= fq;
			jz -= fq;
			m8 -= fq;
			m9 += fq;
			m10 += fq;
			m11 -= fq;
			m12 -= fq;
			m15 += fq;
			m16 += fq;
			m18 -= fq;

			// q=13
			nread = neighborList[n+12*Np];
			fq = dist[nread];
			//fq = dist[16*Np+n];
			rho += fq;
			m1 += 8.0*fq;
			m2 += fq;
			jx += fq;
			m4 += fq;
			jz -= fq;
			m8 -= fq;
			m9 += fq;
			m10 += fq;
			m11 -= fq;
			m12 -= fq;
			m15 -= fq;
			m16 -= fq;
			m18 -= fq;

			// q=14
			nread = neighborList[n+13*Np];
			fq = dist[nread];
			//fq = dist[7*Np+n];
			rho += fq;
			m1 += 8.0*fq;
			m2 += fq;
			jx -= fq;
			m4 -= fq;
			jz += fq;
			m8 += fq;
			m9 += fq;
			m10 += fq;
			m11 -= fq;
			m12 -= fq;
			m15 -= fq;
			m16 += fq;
			m18 += fq;

			// q=15
			nread = neighborList[n+14*Np];
			fq = dist[nread];
			//fq = dist[17*Np+n];
			rho += fq;
			m1 += 8.0*fq;
			m2 += fq;
			jy += fq;
			m6 += fq;
			jz += fq;
			m8 += fq;
			m9 -= 2.0*fq;
			m10 -= 2.0*fq;
			m14 = fq;
			m17 += fq;
			m18 -= fq;

			// q=16
			nread = neighborList[n+15*Np];
			fq = dist[nread];
			//fq = dist[8*Np+n];
			rho += fq;
			m1 += 8.0*fq;
			m2 += fq;
			jy -= fq;
			m6 -= fq;
			jz -= fq;
			m8 -= fq;
			m9 -= 2.0*fq;
			m10 -= 2.0*fq;
			m14 += fq;
			m17 -= fq;
			m18 += fq;

			// q=17
			//fq = dist[18*Np+n];
			nread = neighborList[n+16*Np];
			fq = dist[nread];
			rho += fq;
			m1 += 8.0*fq;
			m2 += fq;
			jy += fq;
			m6 += fq;
			jz -= fq;
			m8 -= fq;
			m9 -= 2.0*fq;
			m10 -= 2.0*fq;
			m14 -= fq;
			m17 += fq;
			m18 += fq;

			// q=18
			nread = neighborList[n+17*Np];
			fq = dist[nread];
			//fq = dist[9*Np+n];
			rho += fq;
			m1 += 8.0*fq;
			m2 += fq;
			jy -= fq;
			m6 -= fq;
			jz += fq;
			m8 += fq;
			m9 -= 2.0*fq;
			m10 -= 2.0*fq;
			m14 -= fq;
			m17 -= fq;
			m18 -= fq;

			//..............incorporate external force................................................
			//..............carry out relaxation process...............................................
			m1 = m1 + rlx_setA*((19*(jx*jx+jy*jy+jz*jz)/rho - 11*rho) - m1);
			m2 = m2 + rlx_setA*((3*rho - 5.5*(jx*jx+jy*jy+jz*jz)/rho) - m2);
			m4 = m4 + rlx_setB*((-0.6666666666666666*jx) - m4);
			m6 = m6 + rlx_setB*((-0.6666666666666666*jy) - m6);
			m8 = m8 + rlx_setB*((-0.6666666666666666*jz) - m8);
			m9 = m9 + rlx_setA*(((2*jx*jx-jy*jy-jz*jz)/rho) - m9);
			m10 = m10 + rlx_setA*(-0.5*((2*jx*jx-jy*jy-jz*jz)/rho) - m10);
			m11 = m11 + rlx_setA*(((jy*jy-jz*jz)/rho) - m11);
			m12 = m12 + rlx_setA*(-0.5*((jy*jy-jz*jz)/rho) - m12);
			m13 = m13 + rlx_setA*((jx*jy/rho) - m13);
			m14 = m14 + rlx_setA*((jy*jz/rho) - m14);
			m15 = m15 + rlx_setA*((jx*jz/rho) - m15);
			m16 = m16 + rlx_setB*( - m16);
			m17 = m17 + rlx_setB*( - m17);
			m18 = m18 + rlx_setB*( - m18);
			//.......................................................................................................
			//.................inverse transformation......................................................

			// q=0
			fq = mrt_V1*rho-mrt_V2*m1+mrt_V3*m2;
			dist[n] = fq;

			// q = 1
			fq = mrt_V1*rho-mrt_V4*m1-mrt_V5*m2+0.1*(jx-m4)+mrt_V6*(m9-m10)+0.16666666*Fx;
			nread = neighborList[n+Np];
			dist[nread] = fq;

			// q=2
			fq = mrt_V1*rho-mrt_V4*m1-mrt_V5*m2+0.1*(m4-jx)+mrt_V6*(m9-m10) -  0.16666666*Fx;
			nread = neighborList[n];
			dist[nread] = fq;

			// q = 3
			fq = mrt_V1*rho-mrt_V4*m1-mrt_V5*m2+0.1*(jy-m6)+mrt_V7*(m10-m9)+mrt_V8*(m11-m12) + 0.16666666*Fy;
			nread = neighborList[n+3*Np];
			dist[nread] = fq;

			// q = 4
			fq = mrt_V1*rho-mrt_V4*m1-mrt_V5*m2+0.1*(m6-jy)+mrt_V7*(m10-m9)+mrt_V8*(m11-m12) - 0.16666666*Fy;
			nread = neighborList[n+2*Np];
			dist[nread] = fq;

			// q = 5
			fq = mrt_V1*rho-mrt_V4*m1-mrt_V5*m2+0.1*(jz-m8)+mrt_V7*(m10-m9)+mrt_V8*(m12-m11) + 0.16666666*Fz;
			nread = neighborList[n+5*Np];
			dist[nread] = fq;

			// q = 6
			fq = mrt_V1*rho-mrt_V4*m1-mrt_V5*m2+0.1*(m8-jz)+mrt_V7*(m10-m9)+mrt_V8*(m12-m11) - 0.16666666*Fz;
			nread = neighborList[n+4*Np];
			dist[nread] = fq;

			// q = 7
			fq = mrt_V1*rho+mrt_V9*m1+mrt_V10*m2+0.1*(jx+jy)+0.025*(m4+m6)+mrt_V7*m9+mrt_V11*m10+
					mrt_V8*m11+mrt_V12*m12+0.25*m13+0.125*(m16-m17) + 0.08333333333*(Fx+Fy);
			
			nread = neighborList[n+7*Np];
			dist[nread] = fq;

			// q = 8
			fq = mrt_V1*rho+mrt_V9*m1+mrt_V10*m2-0.1*(jx+jy)-0.025*(m4+m6) +mrt_V7*m9+mrt_V11*m10+mrt_V8*m11
					+mrt_V12*m12+0.25*m13+0.125*(m17-m16) - 0.08333333333*(Fx+Fy);
			nread = neighborList[n+6*Np];
			dist[nread] = fq;

			// q = 9
			fq = mrt_V1*rho+mrt_V9*m1+mrt_V10*m2+0.1*(jx-jy)+0.025*(m4-m6)+mrt_V7*m9+mrt_V11*m10+
					mrt_V8*m11+mrt_V12*m12-0.25*m13+0.125*(m16+m17) + 0.08333333333*(Fx-Fy);
			nread = neighborList[n+9*Np];
			dist[nread] = fq;

			// q = 10
			fq = mrt_V1*rho+mrt_V9*m1+mrt_V10*m2+0.1*(jy-jx)+0.025*(m6-m4)+mrt_V7*m9+mrt_V11*m10+
					mrt_V8*m11+mrt_V12*m12-0.25*m13-0.125*(m16+m17)- 0.08333333333*(Fx-Fy);
			nread = neighborList[n+8*Np];
			dist[nread] = fq;

			// q = 11
			fq = mrt_V1*rho+mrt_V9*m1
					+mrt_V10*m2+0.1*(jx+jz)+0.025*(m4+m8)
					+mrt_V7*m9+mrt_V11*m10-mrt_V8*m11
					-mrt_V12*m12+0.25*m15+0.125*(m18-m16) + 0.08333333333*(Fx+Fz);
			nread = neighborList[n+11*Np];
			dist[nread] = fq;

			// q = 12
			fq = mrt_V1*rho+mrt_V9*m1+mrt_V10*m2-0.1*(jx+jz)-0.025*(m4+m8)+
					mrt_V7*m9+mrt_V11*m10-mrt_V8*m11-mrt_V12*m12+0.25*m15+0.125*(m16-m18) - 0.08333333333*(Fx+Fz);
			nread = neighborList[n+10*Np];
			dist[nread]= fq;

			// q = 13
			fq = mrt_V1*rho+mrt_V9*m1
					+mrt_V10*m2+0.1*(jx-jz)+0.025*(m4-m8)
					+mrt_V7*m9+mrt_V11*m10-mrt_V8*m11
					-mrt_V12*m12-0.25*m15-0.125*(m16+m18) + 0.08333333333*(Fx-Fz);
			nread = neighborList[n+13*Np];
			dist[nread] = fq;

			// q= 14
			fq = mrt_V1*rho+mrt_V9*m1
					+mrt_V10*m2+0.1*(jz-jx)+0.025*(m8-m4)
					+mrt_V7*m9+mrt_V11*m10-mrt_V8*m11
					-mrt_V12*m12-0.25*m15+0.125*(m16+m18) - 0.08333333333*(Fx-Fz);
			nread = neighborList[n+12*Np];
			dist[nread] = fq;


			// q = 15
			fq = mrt_V1*rho+mrt_V9*m1
					+mrt_V10*m2+0.1*(jy+jz)+0.025*(m6+m8)
					-mrt_V6*m9-mrt_V7*m10+0.25*m14+0.125*(m17-m18) + 0.08333333333*(Fy+Fz);
			nread = neighborList[n+15*Np];
			dist[nread] = fq;

			// q = 16
			fq =  mrt_V1*rho+mrt_V9*m1
					+mrt_V10*m2-0.1*(jy+jz)-0.025*(m6+m8)
					-mrt_V6*m9-mrt_V7*m10+0.25*m14+0.125*(m18-m17)- 0.08333333333*(Fy+Fz);
			nread = neighborList[n+14*Np];
			dist[nread] = fq;


			// q = 17
			fq = mrt_V1*rho+mrt_V9*m1
					+mrt_V10*m2+0.1*(jy-jz)+0.025*(m6-m8)
					-mrt_V6*m9-mrt_V7*m10-0.25*m14+0.125*(m17+m18) + 0.08333333333*(Fy-Fz);
			nread = neighborList[n+17*Np];
			dist[nread] = fq;

			// q = 18
			fq = mrt_V1*rho+mrt_V9*m1
					+mrt_V10*m2+0.1*(jz-jy)+0.025*(m8-m6)
					-mrt_V6*m9-mrt_V7*m10-0.25*m14-0.125*(m17+m18) - 0.08333333333*(Fy-Fz);
			nread = neighborList[n+16*Np];
			dist[nread] = fq;

		}
	}
}


//__launch_bounds__(512,1)
__global__ void 
dvc_ScaLBL_AAeven_MRT(double *dist, int start, int finish, int Np, double rlx_setA, double rlx_setB, double Fx, double Fy, double Fz) {

	int n;
	double fq;
	// conserved momemnts
	double rho,jx,jy,jz;
	// non-conserved moments
	double m1,m2,m4,m6,m8,m9,m10,m11,m12,m13,m14,m15,m16,m17,m18;
	int S = Np/NBLOCKS/NTHREADS + 1;
	for (int s=0; s<S; s++){
		//........Get 1-D index for this thread....................
		n = S*blockIdx.x*blockDim.x + s*blockDim.x + threadIdx.x + start;

		if ( n<finish ){

			// q=0
			fq = dist[n];
			rho = fq;
			m1  = -30.0*fq;
			m2  = 12.0*fq;

			// q=1
			fq = dist[2*Np+n];
			rho += fq;
			m1 -= 11.0*fq;
			m2 -= 4.0*fq;
			jx = fq;
			m4 = -4.0*fq;
			m9 = 2.0*fq;
			m10 = -4.0*fq;

			// q=2
			fq = dist[1*Np+n];
			rho += fq;
			m1 -= 11.0*(fq);
			m2 -= 4.0*(fq);
			jx -= fq;
			m4 += 4.0*(fq);
			m9 += 2.0*(fq);
			m10 -= 4.0*(fq);

			// q=3
			fq = dist[4*Np+n];
			rho += fq;
			m1 -= 11.0*fq;
			m2 -= 4.0*fq;
			jy = fq;
			m6 = -4.0*fq;
			m9 -= fq;
			m10 += 2.0*fq;
			m11 = fq;
			m12 = -2.0*fq;

			// q = 4
			fq = dist[3*Np+n];
			rho+= fq;
			m1 -= 11.0*fq;
			m2 -= 4.0*fq;
			jy -= fq;
			m6 += 4.0*fq;
			m9 -= fq;
			m10 += 2.0*fq;
			m11 += fq;
			m12 -= 2.0*fq;

			// q=5
			fq = dist[6*Np+n];
			rho += fq;
			m1 -= 11.0*fq;
			m2 -= 4.0*fq;
			jz = fq;
			m8 = -4.0*fq;
			m9 -= fq;
			m10 += 2.0*fq;
			m11 -= fq;
			m12 += 2.0*fq;

			// q = 6
			fq = dist[5*Np+n];
			rho+= fq;
			m1 -= 11.0*fq;
			m2 -= 4.0*fq;
			jz -= fq;
			m8 += 4.0*fq;
			m9 -= fq;
			m10 += 2.0*fq;
			m11 -= fq;
			m12 += 2.0*fq;

			// q=7
			fq = dist[8*Np+n];
			rho += fq;
			m1 += 8.0*fq;
			m2 += fq;
			jx += fq;
			m4 += fq;
			jy += fq;
			m6 += fq;
			m9  += fq;
			m10 += fq;
			m11 += fq;
			m12 += fq;
			m13 = fq;
			m16 = fq;
			m17 = -fq;

			// q = 8
			fq = dist[7*Np+n];
			rho += fq;
			m1 += 8.0*fq;
			m2 += fq;
			jx -= fq;
			m4 -= fq;
			jy -= fq;
			m6 -= fq;
			m9 += fq;
			m10 += fq;
			m11 += fq;
			m12 += fq;
			m13 += fq;
			m16 -= fq;
			m17 += fq;

			// q=9
			fq = dist[10*Np+n];
			rho += fq;
			m1 += 8.0*fq;
			m2 += fq;
			jx += fq;
			m4 += fq;
			jy -= fq;
			m6 -= fq;
			m9 += fq;
			m10 += fq;
			m11 += fq;
			m12 += fq;
			m13 -= fq;
			m16 += fq;
			m17 += fq;

			// q = 10
			fq = dist[9*Np+n];
			rho += fq;
			m1 += 8.0*fq;
			m2 += fq;
			jx -= fq;
			m4 -= fq;
			jy += fq;
			m6 += fq;
			m9 += fq;
			m10 += fq;
			m11 += fq;
			m12 += fq;
			m13 -= fq;
			m16 -= fq;
			m17 -= fq;

			// q=11
			fq = dist[12*Np+n];
			rho += fq;
			m1 += 8.0*fq;
			m2 += fq;
			jx += fq;
			m4 += fq;
			jz += fq;
			m8 += fq;
			m9 += fq;
			m10 += fq;
			m11 -= fq;
			m12 -= fq;
			m15 = fq;
			m16 -= fq;
			m18 = fq;

			// q=12
			fq = dist[11*Np+n];
			rho += fq;
			m1 += 8.0*fq;
			m2 += fq;
			jx -= fq;
			m4 -= fq;
			jz -= fq;
			m8 -= fq;
			m9 += fq;
			m10 += fq;
			m11 -= fq;
			m12 -= fq;
			m15 += fq;
			m16 += fq;
			m18 -= fq;

			// q=13
			fq = dist[14*Np+n];
			rho += fq;
			m1 += 8.0*fq;
			m2 += fq;
			jx += fq;
			m4 += fq;
			jz -= fq;
			m8 -= fq;
			m9 += fq;
			m10 += fq;
			m11 -= fq;
			m12 -= fq;
			m15 -= fq;
			m16 -= fq;
			m18 -= fq;

			// q=14
			fq = dist[13*Np+n];
			rho += fq;
			m1 += 8.0*fq;
			m2 += fq;
			jx -= fq;
			m4 -= fq;
			jz += fq;
			m8 += fq;
			m9 += fq;
			m10 += fq;
			m11 -= fq;
			m12 -= fq;
			m15 -= fq;
			m16 += fq;
			m18 += fq;

			// q=15
			fq = dist[16*Np+n];
			rho += fq;
			m1 += 8.0*fq;
			m2 += fq;
			jy += fq;
			m6 += fq;
			jz += fq;
			m8 += fq;
			m9 -= 2.0*fq;
			m10 -= 2.0*fq;
			m14 = fq;
			m17 += fq;
			m18 -= fq;

			// q=16
			fq = dist[15*Np+n];
			rho += fq;
			m1 += 8.0*fq;
			m2 += fq;
			jy -= fq;
			m6 -= fq;
			jz -= fq;
			m8 -= fq;
			m9 -= 2.0*fq;
			m10 -= 2.0*fq;
			m14 += fq;
			m17 -= fq;
			m18 += fq;

			// q=17
			fq = dist[18*Np+n];
			rho += fq;
			m1 += 8.0*fq;
			m2 += fq;
			jy += fq;
			m6 += fq;
			jz -= fq;
			m8 -= fq;
			m9 -= 2.0*fq;
			m10 -= 2.0*fq;
			m14 -= fq;
			m17 += fq;
			m18 += fq;

			// q=18
			fq = dist[17*Np+n];
			rho += fq;
			m1 += 8.0*fq;
			m2 += fq;
			jy -= fq;
			m6 -= fq;
			jz += fq;
			m8 += fq;
			m9 -= 2.0*fq;
			m10 -= 2.0*fq;
			m14 -= fq;
			m17 -= fq;
			m18 -= fq;

			//........................................................................
			//					READ THE DISTRIBUTIONS
			//		(read from opposite array due to previous swap operation)
			//........................................................................

			//..............incorporate external force................................................
			//..............carry out relaxation process...............................................
			m1 = m1 + rlx_setA*((19*(jx*jx+jy*jy+jz*jz)/rho - 11*rho) - m1);
			m2 = m2 + rlx_setA*((3*rho - 5.5*(jx*jx+jy*jy+jz*jz)/rho) - m2);
			m4 = m4 + rlx_setB*((-0.6666666666666666*jx) - m4);
			m6 = m6 + rlx_setB*((-0.6666666666666666*jy) - m6);
			m8 = m8 + rlx_setB*((-0.6666666666666666*jz) - m8);
			m9 = m9 + rlx_setA*(((2*jx*jx-jy*jy-jz*jz)/rho) - m9);
			m10 = m10 + rlx_setA*(-0.5*((2*jx*jx-jy*jy-jz*jz)/rho) - m10);
			m11 = m11 + rlx_setA*(((jy*jy-jz*jz)/rho) - m11);
			m12 = m12 + rlx_setA*(-0.5*((jy*jy-jz*jz)/rho) - m12);
			m13 = m13 + rlx_setA*((jx*jy/rho) - m13);
			m14 = m14 + rlx_setA*((jy*jz/rho) - m14);
			m15 = m15 + rlx_setA*((jx*jz/rho) - m15);
			m16 = m16 + rlx_setB*( - m16);
			m17 = m17 + rlx_setB*( - m17);
			m18 = m18 + rlx_setB*( - m18);
			//.......................................................................................................
			//.................inverse transformation......................................................

			// q=0
			fq = mrt_V1*rho-mrt_V2*m1+mrt_V3*m2;
			dist[n] = fq;

			// q = 1
			fq = mrt_V1*rho-mrt_V4*m1-mrt_V5*m2+0.1*(jx-m4)+mrt_V6*(m9-m10) + 0.16666666*Fx;
			dist[1*Np+n] = fq;

			// q=2
			fq = mrt_V1*rho-mrt_V4*m1-mrt_V5*m2+0.1*(m4-jx)+mrt_V6*(m9-m10) -  0.16666666*Fx;
			dist[2*Np+n] = fq;

			// q = 3
			fq = mrt_V1*rho-mrt_V4*m1-mrt_V5*m2+0.1*(jy-m6)+mrt_V7*(m10-m9)+mrt_V8*(m11-m12) + 0.16666666*Fy;
			dist[3*Np+n] = fq;

			// q = 4
			fq = mrt_V1*rho-mrt_V4*m1-mrt_V5*m2+0.1*(m6-jy)+mrt_V7*(m10-m9)+mrt_V8*(m11-m12) - 0.16666666*Fy;
			dist[4*Np+n] = fq;

			// q = 5
			fq = mrt_V1*rho-mrt_V4*m1-mrt_V5*m2+0.1*(jz-m8)+mrt_V7*(m10-m9)+mrt_V8*(m12-m11) + 0.16666666*Fz;
			dist[5*Np+n] = fq;

			// q = 6
			fq = mrt_V1*rho-mrt_V4*m1-mrt_V5*m2+0.1*(m8-jz)+mrt_V7*(m10-m9)+mrt_V8*(m12-m11) - 0.16666666*Fz;
			dist[6*Np+n] = fq;

			// q = 7
			fq = mrt_V1*rho+mrt_V9*m1+mrt_V10*m2+0.1*(jx+jy)+0.025*(m4+m6)+
					mrt_V7*m9+mrt_V11*m10+mrt_V8*m11+mrt_V12*m12+0.25*m13+0.125*(m16-m17) + 
					0.08333333333*(Fx+Fy);
			dist[7*Np+n] = fq;


			// q = 8
			fq = mrt_V1*rho+mrt_V9*m1+mrt_V10*m2-0.1*(jx+jy)-0.025*(m4+m6) +mrt_V7*m9+mrt_V11*m10+mrt_V8*m11
					+mrt_V12*m12+0.25*m13+0.125*(m17-m16) - 0.08333333333*(Fx+Fy);
			dist[8*Np+n] = fq;

			// q = 9
			fq = mrt_V1*rho+mrt_V9*m1+mrt_V10*m2+0.1*(jx-jy)+0.025*(m4-m6)+
					mrt_V7*m9+mrt_V11*m10+mrt_V8*m11+mrt_V12*m12-0.25*m13+0.125*(m16+m17)+
					0.08333333333*(Fx-Fy);
			dist[9*Np+n] = fq;

			// q = 10
			fq = mrt_V1*rho+mrt_V9*m1+mrt_V10*m2+0.1*(jy-jx)+0.025*(m6-m4)+
					mrt_V7*m9+mrt_V11*m10+mrt_V8*m11+mrt_V12*m12-0.25*m13-0.125*(m16+m17)-
					0.08333333333*(Fx-Fy);
			dist[10*Np+n] = fq;


			// q = 11
			fq = mrt_V1*rho+mrt_V9*m1
					+mrt_V10*m2+0.1*(jx+jz)+0.025*(m4+m8)
					+mrt_V7*m9+mrt_V11*m10-mrt_V8*m11
					-mrt_V12*m12+0.25*m15+0.125*(m18-m16) + 0.08333333333*(Fx+Fz);
			dist[11*Np+n] = fq;

			// q = 12
			fq = mrt_V1*rho+mrt_V9*m1+mrt_V10*m2-0.1*(jx+jz)-0.025*(m4+m8)+
					mrt_V7*m9+mrt_V11*m10-mrt_V8*m11-mrt_V12*m12+0.25*m15+0.125*(m16-m18)-
					0.08333333333*(Fx+Fz);
			dist[12*Np+n] = fq;

			// q = 13
			fq = mrt_V1*rho+mrt_V9*m1
					+mrt_V10*m2+0.1*(jx-jz)+0.025*(m4-m8)
					+mrt_V7*m9+mrt_V11*m10-mrt_V8*m11
					-mrt_V12*m12-0.25*m15-0.125*(m16+m18) + 0.08333333333*(Fx-Fz);
			dist[13*Np+n] = fq;

			// q= 14
			fq = mrt_V1*rho+mrt_V9*m1
					+mrt_V10*m2+0.1*(jz-jx)+0.025*(m8-m4)
					+mrt_V7*m9+mrt_V11*m10-mrt_V8*m11
					-mrt_V12*m12-0.25*m15+0.125*(m16+m18) - 0.08333333333*(Fx-Fz);

			dist[14*Np+n] = fq;

			// q = 15
			fq = mrt_V1*rho+mrt_V9*m1
					+mrt_V10*m2+0.1*(jy+jz)+0.025*(m6+m8)
					-mrt_V6*m9-mrt_V7*m10+0.25*m14+0.125*(m17-m18) + 0.08333333333*(Fy+Fz);
			dist[15*Np+n] = fq;

			// q = 16
			fq =  mrt_V1*rho+mrt_V9*m1
					+mrt_V10*m2-0.1*(jy+jz)-0.025*(m6+m8)
					-mrt_V6*m9-mrt_V7*m10+0.25*m14+0.125*(m18-m17)- 0.08333333333*(Fy+Fz);
			dist[16*Np+n] = fq;


			// q = 17
			fq = mrt_V1*rho+mrt_V9*m1
					+mrt_V10*m2+0.1*(jy-jz)+0.025*(m6-m8)
					-mrt_V6*m9-mrt_V7*m10-0.25*m14+0.125*(m17+m18) + 0.08333333333*(Fy-Fz);
			dist[17*Np+n] = fq;

			// q = 18
			fq = mrt_V1*rho+mrt_V9*m1
					+mrt_V10*m2+0.1*(jz-jy)+0.025*(m8-m6)
					-mrt_V6*m9-mrt_V7*m10-0.25*m14-0.125*(m17+m18) - 0.08333333333*(Fy-Fz);
			dist[18*Np+n] = fq;
			//........................................................................
		}
	}
}

__global__  void dvc_ScaLBL_D3Q19_Momentum(double *dist, double *vel, int N)
{
	int n;
	// distributions
	double f1,f2,f3,f4,f5,f6,f7,f8,f9;
	double f10,f11,f12,f13,f14,f15,f16,f17,f18;
	double vx,vy,vz;

	int S = N/NBLOCKS/NTHREADS + 1;
	for (int s=0; s<S; s++){
		//........Get 1-D index for this thread....................
		n = S*blockIdx.x*blockDim.x + s*blockDim.x + threadIdx.x;
		if (n<N){

			f2 = dist[2*N+n];
			f4 = dist[4*N+n];
			f6 = dist[6*N+n];
			f8 = dist[8*N+n];
			f10 = dist[10*N+n];
			f12 = dist[12*N+n];
			f14 = dist[14*N+n];
			f16 = dist[16*N+n];
			f18 = dist[18*N+n];
			//........................................................................
			f1 = dist[N+n];
			f3 = dist[3*N+n];
			f5 = dist[5*N+n];
			f7 = dist[7*N+n];
			f9 = dist[9*N+n];
			f11 = dist[11*N+n];
			f13 = dist[13*N+n];
			f15 = dist[15*N+n];
			f17 = dist[17*N+n];			

			//.................Compute the velocity...................................
			vx = f1-f2+f7-f8+f9-f10+f11-f12+f13-f14;
			vy = f3-f4+f7-f8-f9+f10+f15-f16+f17-f18;
			vz = f5-f6+f11-f12-f13+f14+f15-f16-f17+f18;
			//..................Write the velocity.....................................
			vel[n] = vx;
			vel[N+n] = vy;
			vel[2*N+n] = vz;
			//........................................................................
		}
	}
}

__global__  void dvc_ScaLBL_D3Q19_Pressure(const double *dist, double *Pressure, int N)
{
	int n;
	// distributions
	double f0,f1,f2,f3,f4,f5,f6,f7,f8,f9;
	double f10,f11,f12,f13,f14,f15,f16,f17,f18;

	int S = N/NBLOCKS/NTHREADS + 1;
	for (int s=0; s<S; s++){
		//........Get 1-D index for this thread....................
		n = S*blockIdx.x*blockDim.x + s*blockDim.x + threadIdx.x;
		if (n<N){				//.......................................................................
			// Registers to store the distributions
			//........................................................................
			//........................................................................
			// Registers to store the distributions
			//........................................................................
			f0 = dist[n];
			f2 = dist[2*N+n];
			f4 = dist[4*N+n];
			f6 = dist[6*N+n];
			f8 = dist[8*N+n];
			f10 = dist[10*N+n];
			f12 = dist[12*N+n];
			f14 = dist[14*N+n];
			f16 = dist[16*N+n];
			f18 = dist[18*N+n];
			//........................................................................
			f1 = dist[N+n];
			f3 = dist[3*N+n];
			f5 = dist[5*N+n];
			f7 = dist[7*N+n];
			f9 = dist[9*N+n];
			f11 = dist[11*N+n];
			f13 = dist[13*N+n];
			f15 = dist[15*N+n];
			f17 = dist[17*N+n];
			//.................Compute the velocity...................................
			Pressure[n] = 0.3333333333333333*(f0+f2+f1+f4+f3+f6+f5+f8+f7+f10+
					f9+f12+f11+f14+f13+f16+f15+f18+f17);
		}
	}
}

__global__  void dvc_ScaLBL_D3Q19_AAeven_Pressure_BC_z(int *list, double *dist, double din, int count, int Np)
{
	int idx, n;
	// distributions
	double f0,f1,f2,f3,f4,f5,f6,f7,f8,f9;
	double f10,f11,f12,f13,f14,f15,f16,f17,f18;
	double ux,uy,uz,Cyz,Cxz;
	ux = uy = 0.0;

	idx = blockIdx.x*blockDim.x + threadIdx.x;

	if (idx < count){

		n = list[idx];
		f0 = dist[n];
		f1 = dist[2*Np+n];
		f2 = dist[1*Np+n];
		f3 = dist[4*Np+n];
		f4 = dist[3*Np+n];
		f6 = dist[5*Np+n];
		f7 = dist[8*Np+n];
		f8 = dist[7*Np+n];
		f9 = dist[10*Np+n];
		f10 = dist[9*Np+n];
		f12 = dist[11*Np+n];
		f13 = dist[14*Np+n];
		f16 = dist[15*Np+n];
		f17 = dist[18*Np+n];
		//...................................................
		// Determine the inlet flow velocity
		//ux = (f1-f2+f7-f8+f9-f10+f11-f12+f13-f14);
		//uy = (f3-f4+f7-f8-f9+f10+f15-f16+f17-f18);
		uz = din - (f0+f1+f2+f3+f4+f7+f8+f9+f10 + 2*(f6+f12+f13+f16+f17));

		Cxz = 0.5*(f1+f7+f9-f2-f10-f8) - 0.3333333333333333*ux;
		Cyz = 0.5*(f3+f7+f10-f4-f9-f8) - 0.3333333333333333*uy;

		f5 = f6 + 0.33333333333333338*uz;
		f11 = f12 + 0.16666666666666678*(uz+ux)-Cxz;
		f14 = f13 + 0.16666666666666678*(uz-ux)+Cxz;
		f15 = f16 + 0.16666666666666678*(uy+uz)-Cyz;
		f18 = f17 + 0.16666666666666678*(uz-uy)+Cyz;
		//........Store in "opposite" memory location..........
		dist[6*Np+n] = f5;
		dist[12*Np+n] = f11;
		dist[13*Np+n] = f14;
		dist[16*Np+n] = f15;
		dist[17*Np+n] = f18;
	}
}

__global__  void dvc_ScaLBL_D3Q19_AAeven_Pressure_BC_Z(int *list, double *dist, double dout, int count, int Np)
{
	int idx,n;
	// distributions
	double f0,f1,f2,f3,f4,f5,f6,f7,f8,f9;
	double f10,f11,f12,f13,f14,f15,f16,f17,f18;
	double ux,uy,uz,Cyz,Cxz;
	ux = uy = 0.0;

	idx = blockIdx.x*blockDim.x + threadIdx.x;

	// Loop over the boundary - threadblocks delineated by start...finish
	if ( idx < count ){

		n = list[idx];
		//........................................................................
		// Read distributions 
		//........................................................................
		f0 = dist[n];
		f1 = dist[2*Np+n];
		f2 = dist[1*Np+n];
		f3 = dist[4*Np+n];
		f4 = dist[3*Np+n];
		f5 = dist[6*Np+n];
		f7 = dist[8*Np+n];
		f8 = dist[7*Np+n];
		f9 = dist[10*Np+n];
		f10 = dist[9*Np+n];
		f11 = dist[12*Np+n];
		f14 = dist[13*Np+n];
		f15 = dist[16*Np+n];
		f18 = dist[17*Np+n];
		
		// Determine the outlet flow velocity
		//ux = f1-f2+f7-f8+f9-f10+f11-f12+f13-f14;
		//uy = f3-f4+f7-f8-f9+f10+f15-f16+f17-f18;
		uz = -dout + (f0+f1+f2+f3+f4+f7+f8+f9+f10 + 2*(f5+f11+f14+f15+f18));

		Cxz = 0.5*(f1+f7+f9-f2-f10-f8) - 0.3333333333333333*ux;
		Cyz = 0.5*(f3+f7+f10-f4-f9-f8) - 0.3333333333333333*uy;

		f6 = f5 - 0.33333333333333338*uz;
		f12 = f11 - 0.16666666666666678*(uz+ux)+Cxz;
		f13 = f14 - 0.16666666666666678*(uz-ux)-Cxz;
		f16 = f15 - 0.16666666666666678*(uy+uz)+Cyz;
		f17 = f18 - 0.16666666666666678*(uz-uy)-Cyz;

		dist[5*Np+n] = f6;
		dist[11*Np+n] = f12;
		dist[14*Np+n] = f13;
		dist[15*Np+n] = f16;
		dist[18*Np+n] = f17;
		//...................................................
	}
}
__global__  void dvc_ScaLBL_D3Q19_Reflection_BC_z(int *list, double *dist, int count, int Np){
	int idx, n;
	idx = blockIdx.x*blockDim.x + threadIdx.x;
	if (idx < count){
		n = list[idx];
		double f5 = 0.111111111111111111111111 - dist[6*Np+n];
		double f11 = 0.05555555555555555555556 - dist[12*Np+n];
		double f14 = 0.05555555555555555555556 - dist[13*Np+n];
		double f15 = 0.05555555555555555555556 - dist[16*Np+n];
		double f18 = 0.05555555555555555555556 - dist[17*Np+n];
		
		dist[6*Np+n] = f5;
		dist[12*Np+n] = f11;
		dist[13*Np+n] = f14;
		dist[16*Np+n] = f15;
		dist[17*Np+n] = f18;
	}
}

__global__  void dvc_ScaLBL_D3Q19_Reflection_BC_Z(int *list, double *dist, int count, int Np){
	int idx, n;
	idx = blockIdx.x*blockDim.x + threadIdx.x;
	if (idx < count){
		n = list[idx];
		double f6 = 0.111111111111111111111111 - dist[5*Np+n];
		double f12 = 0.05555555555555555555556 - dist[11*Np+n];
		double f13 = 0.05555555555555555555556 - dist[14*Np+n] ;
		double f16 = 0.05555555555555555555556 - dist[15*Np+n];
		double f17 = 0.05555555555555555555556 - dist[18*Np+n];
		
		dist[5*Np+n] = f6;
		dist[11*Np+n] = f12;
		dist[14*Np+n] = f13;
		dist[15*Np+n] = f16;
		dist[18*Np+n] = f17;
	}
}

__global__  void dvc_ScaLBL_D3Q19_AAodd_Pressure_BC_z(int *d_neighborList, int *list, double *dist, double din, int count, int Np)
{
	int idx, n;
	int nread;
	int nr5,nr11,nr14,nr15,nr18;
	// distributions
	double f0,f1,f2,f3,f4,f5,f6,f7,f8,f9;
	double f10,f11,f12,f13,f14,f15,f16,f17,f18;
	double ux,uy,uz,Cyz,Cxz;
	ux = uy = 0.0;

	idx = blockIdx.x*blockDim.x + threadIdx.x;

	if (idx < count){
		
		n = list[idx];
		f0 = dist[n];
				
		nread = d_neighborList[n];
		f1 = dist[nread];

		nread = d_neighborList[n+2*Np];
		f3 = dist[nread];

		nread = d_neighborList[n+6*Np];
		f7 = dist[nread];

		nread = d_neighborList[n+8*Np];
		f9 = dist[nread];

		nread = d_neighborList[n+12*Np];
		f13 = dist[nread];

		nread = d_neighborList[n+16*Np];
		f17 = dist[nread];

		nread = d_neighborList[n+Np];
		f2 = dist[nread];

		nread = d_neighborList[n+3*Np];
		f4 = dist[nread];

		nread = d_neighborList[n+5*Np];
		f6 = dist[nread];

		nread = d_neighborList[n+7*Np];
		f8 = dist[nread];

		nread = d_neighborList[n+9*Np];
		f10 = dist[nread];

		nread = d_neighborList[n+11*Np];
		f12 = dist[nread];

		nread = d_neighborList[n+15*Np];
		f16 = dist[nread];

		// Unknown distributions
		nr5 = d_neighborList[n+4*Np];
		nr11 = d_neighborList[n+10*Np];
		nr15 = d_neighborList[n+14*Np];
		nr14 = d_neighborList[n+13*Np];
		nr18 = d_neighborList[n+17*Np];
		
		//...................................................
		//........Determine the inlet flow velocity.........
		//ux = (f1-f2+f7-f8+f9-f10+f11-f12+f13-f14);
		//uy = (f3-f4+f7-f8-f9+f10+f15-f16+f17-f18);
		uz = din - (f0+f1+f2+f3+f4+f7+f8+f9+f10 + 2*(f6+f12+f13+f16+f17));

		Cxz = 0.5*(f1+f7+f9-f2-f10-f8) - 0.3333333333333333*ux;
		Cyz = 0.5*(f3+f7+f10-f4-f9-f8) - 0.3333333333333333*uy;

		f5 = f6 + 0.33333333333333338*uz;
		f11 = f12 + 0.16666666666666678*(uz+ux)-Cxz;
		f14 = f13 + 0.16666666666666678*(uz-ux)+Cxz;
		f15 = f16 + 0.16666666666666678*(uy+uz)-Cyz;
		f18 = f17 + 0.16666666666666678*(uz-uy)+Cyz;
		//........Store in "opposite" memory location..........
		dist[nr5] = f5;
		dist[nr11] = f11;
		dist[nr14] = f14;
		dist[nr15] = f15;
		dist[nr18] = f18;
	}
}

__global__  void dvc_ScaLBL_D3Q19_AAodd_Pressure_BC_Z(int *d_neighborList, int *list, double *dist, double dout, int count, int Np)
{
	int idx,n,nread;
	int nr6,nr12,nr13,nr16,nr17;
	// distributions
	double f0,f1,f2,f3,f4,f5,f6,f7,f8,f9;
	double f10,f11,f12,f13,f14,f15,f16,f17,f18;
	double ux,uy,uz,Cyz,Cxz;
	ux = uy = 0.0;

	idx = blockIdx.x*blockDim.x + threadIdx.x;

	// Loop over the boundary - threadblocks delineated by start...finish
	if ( idx < count ){

		n = list[idx];
		//........................................................................
		// Read distributions 
		//........................................................................
		f0 = dist[n];

		nread = d_neighborList[n];
		f1 = dist[nread];

		nread = d_neighborList[n+2*Np];
		f3 = dist[nread];

		nread = d_neighborList[n+4*Np];
		f5 = dist[nread];

		nread = d_neighborList[n+6*Np];
		f7 = dist[nread];

		nread = d_neighborList[n+8*Np];
		f9 = dist[nread];

		nread = d_neighborList[n+10*Np];
		f11 = dist[nread];

		nread = d_neighborList[n+14*Np];
		f15 = dist[nread];


		nread = d_neighborList[n+Np];
		f2 = dist[nread];

		nread = d_neighborList[n+3*Np];
		f4 = dist[nread];

		nread = d_neighborList[n+7*Np];
		f8 = dist[nread];

		nread = d_neighborList[n+9*Np];
		f10 = dist[nread];

		nread = d_neighborList[n+13*Np];
		f14 = dist[nread];

		nread = d_neighborList[n+17*Np];
		f18 = dist[nread];
		
		// unknown distributions
		nr6 = d_neighborList[n+5*Np];
		nr12 = d_neighborList[n+11*Np];
		nr16 = d_neighborList[n+15*Np];
		nr17 = d_neighborList[n+16*Np];
		nr13 = d_neighborList[n+12*Np];

		
		//........Determine the outlet flow velocity.........
		//ux = f1-f2+f7-f8+f9-f10+f11-f12+f13-f14;
		//uy = f3-f4+f7-f8-f9+f10+f15-f16+f17-f18;
		uz = -dout + (f0+f1+f2+f3+f4+f7+f8+f9+f10 + 2*(f5+f11+f14+f15+f18));

		Cxz = 0.5*(f1+f7+f9-f2-f10-f8) - 0.3333333333333333*ux;
		Cyz = 0.5*(f3+f7+f10-f4-f9-f8) - 0.3333333333333333*uy;

		f6 = f5 - 0.33333333333333338*uz;
		f12 = f11 - 0.16666666666666678*(uz+ux)+Cxz;
		f13 = f14 - 0.16666666666666678*(uz-ux)-Cxz;
		f16 = f15 - 0.16666666666666678*(uy+uz)+Cyz;
		f17 = f18 - 0.16666666666666678*(uz-uy)-Cyz;

		//........Store in "opposite" memory location..........
		dist[nr6] = f6;
		dist[nr12] = f12;
		dist[nr13] = f13;
		dist[nr16] = f16;
		dist[nr17] = f17;
		//...................................................
	}
}


__global__  void dvc_ScaLBL_D3Q19_AAeven_Flux_BC_z(int *list, double *dist, double flux, double Area, 
		double *dvcsum, int count, int Np)
{
	int idx, n;
	// distributions
	double f0,f1,f2,f3,f4,f5,f6,f7,f8,f9;
	double f10,f11,f12,f13,f14,f15,f16,f17,f18;
	double factor = 1.f/(Area);
	double sum = 0.f;

	idx = blockIdx.x*blockDim.x + threadIdx.x;

	if (idx < count){
		
		n = list[idx];
		f0 = dist[n];
		f1 = dist[2*Np+n];
		f2 = dist[1*Np+n];
		f3 = dist[4*Np+n];
		f4 = dist[3*Np+n];
		f6 = dist[5*Np+n];
		f7 = dist[8*Np+n];
		f8 = dist[7*Np+n];
		f9 = dist[10*Np+n];
		f10 = dist[9*Np+n];
		f12 = dist[11*Np+n];
		f13 = dist[14*Np+n];
		f16 = dist[15*Np+n];
		f17 = dist[18*Np+n];
		sum = factor*(f0+f1+f2+f3+f4+f7+f8+f9+f10 + 2*(f6+f12+f13+f16+f17));
	}

	//sum = blockReduceSum(sum);
	//if (threadIdx.x==0)
	//   atomicAdd(dvcsum, sum);
	
    extern __shared__ double temp[];
    thread_group g = this_thread_block();
    double block_sum = reduce_sum(g, temp, sum);

    if (g.thread_rank() == 0) atomicAdd(dvcsum, block_sum);
}


__global__  void dvc_ScaLBL_D3Q19_AAodd_Flux_BC_z(int *d_neighborList, int *list, double *dist, double flux, 
		double Area, double *dvcsum, int count, int Np)
{
	int idx, n;
	int nread;

	// distributions
	double f0,f1,f2,f3,f4,f5,f6,f7,f8,f9;
	double f10,f11,f12,f13,f14,f15,f16,f17,f18;
	double factor = 1.f/(Area);
	double sum = 0.f;

	idx = blockIdx.x*blockDim.x + threadIdx.x;

	if (idx < count){
		
		n = list[idx];
				
		f0 = dist[n];
		
		nread = d_neighborList[n];
		f1 = dist[nread];

		nread = d_neighborList[n+2*Np];
		f3 = dist[nread];

		nread = d_neighborList[n+6*Np];
		f7 = dist[nread];

		nread = d_neighborList[n+8*Np];
		f9 = dist[nread];

		nread = d_neighborList[n+12*Np];
		f13 = dist[nread];

		nread = d_neighborList[n+16*Np];
		f17 = dist[nread];

		nread = d_neighborList[n+Np];
		f2 = dist[nread];

		nread = d_neighborList[n+3*Np];
		f4 = dist[nread];

		nread = d_neighborList[n+5*Np];
		f6 = dist[nread];

		nread = d_neighborList[n+7*Np];
		f8 = dist[nread];

		nread = d_neighborList[n+9*Np];
		f10 = dist[nread];

		nread = d_neighborList[n+11*Np];
		f12 = dist[nread];

		nread = d_neighborList[n+15*Np];
		f16 = dist[nread];

		sum = factor*(f0+f1+f2+f3+f4+f7+f8+f9+f10 + 2*(f6+f12+f13+f16+f17));

	}

	//sum = blockReduceSum(sum);
	//if (threadIdx.x==0)
	//   atomicAdd(dvcsum, sum);
	
    extern __shared__ double temp[];
    thread_group g = this_thread_block();
    double block_sum = reduce_sum(g, temp, sum);

    if (g.thread_rank() == 0) atomicAdd(dvcsum, block_sum);
}



extern "C" void ScaLBL_D3Q19_Pack(int q, int *list, int start, int count, double *sendbuf, double *dist, int N){
	int GRID = count / 512 + 1;
	dvc_ScaLBL_D3Q19_Pack <<<GRID,512 >>>(q, list, start, count, sendbuf, dist, N);
}

extern "C" void ScaLBL_D3Q19_Unpack(int q, int *list,  int start, int count, double *recvbuf, double *dist, int N){
	int GRID = count / 512 + 1;
	dvc_ScaLBL_D3Q19_Unpack <<<GRID,512 >>>(q, list, start, count, recvbuf, dist, N);
}
//*************************************************************************


extern "C" void ScaLBL_D3Q19_Init(double *dist, int Np){
	dvc_ScaLBL_D3Q19_Init<<<NBLOCKS,NTHREADS >>>(dist, Np);
	hipError_t err = hipGetLastError();
	if (hipSuccess != err){
		printf("CUDA error in ScaLBL_D3Q19_Init: %s \n",hipGetErrorString(err));
	}
}

extern "C" void ScaLBL_D3Q19_Momentum(double *dist, double *vel, int Np){

	dvc_ScaLBL_D3Q19_Momentum<<<NBLOCKS,NTHREADS >>>(dist, vel, Np);

	hipError_t err = hipGetLastError();
	if (hipSuccess != err){
		printf("CUDA error in ScaLBL_D3Q19_Velocity: %s \n",hipGetErrorString(err));
	}
}

extern "C" void ScaLBL_D3Q19_Pressure(double *fq, double *Pressure, int Np){
	dvc_ScaLBL_D3Q19_Pressure<<< NBLOCKS,NTHREADS >>>(fq, Pressure, Np);
}



extern "C" void ScaLBL_D3Q19_AAeven_Pressure_BC_z(int *list, double *dist, double din, int count, int N){
	int GRID = count / 512 + 1;
	dvc_ScaLBL_D3Q19_AAeven_Pressure_BC_z<<<GRID,512>>>(list, dist, din, count, N);
	hipError_t err = hipGetLastError();
	if (hipSuccess != err){
		printf("CUDA error in ScaLBL_D3Q19_AAeven_Pressure_BC_z (kernel): %s \n",hipGetErrorString(err));
	}
}

extern "C" void ScaLBL_D3Q19_AAeven_Pressure_BC_Z(int *list, double *dist, double dout, int count, int N){
	int GRID = count / 512 + 1;
	dvc_ScaLBL_D3Q19_AAeven_Pressure_BC_Z<<<GRID,512>>>(list, dist, dout, count, N);
	hipError_t err = hipGetLastError();
	if (hipSuccess != err){
		printf("CUDA error in ScaLBL_D3Q19_AAeven_Pressure_BC_Z (kernel): %s \n",hipGetErrorString(err));
	}
}

extern "C" void ScaLBL_D3Q19_AAodd_Pressure_BC_z(int *neighborList, int *list, double *dist, double din, int count, int N){
	int GRID = count / 512 + 1;
	dvc_ScaLBL_D3Q19_AAodd_Pressure_BC_z<<<GRID,512>>>(neighborList, list, dist, din, count, N);
	hipError_t err = hipGetLastError();
	if (hipSuccess != err){
		printf("CUDA error in ScaLBL_D3Q19_AAodd_Pressure_BC_z (kernel): %s \n",hipGetErrorString(err));
	}
}

extern "C" void ScaLBL_D3Q19_AAodd_Pressure_BC_Z(int *neighborList, int *list, double *dist, double dout, int count, int N){
	int GRID = count / 512 + 1;
	dvc_ScaLBL_D3Q19_AAodd_Pressure_BC_Z<<<GRID,512>>>(neighborList, list, dist, dout, count, N);
	hipError_t err = hipGetLastError();
	if (hipSuccess != err){
		printf("CUDA error in ScaLBL_D3Q19_AAodd_Pressure_BC_Z (kernel): %s \n",hipGetErrorString(err));
	}
}


extern "C" double ScaLBL_D3Q19_AAeven_Flux_BC_z(int *list, double *dist, double flux, double area, 
		 int count, int N){

	int GRID = count / 512 + 1;

	// IMPORTANT -- this routine may fail if Nx*Ny > 512*512
	if (count > 512*512){
		printf("WARNING (ScaLBL_D3Q19_Flux_BC_Z): CUDA reduction operation may fail if count > 512*512");
	}

	// Allocate memory to store the sums
	double din;
	double sum[1];
 	double *dvcsum;
	hipMalloc((void **)&dvcsum,sizeof(double)*count);
	hipMemset(dvcsum,0,sizeof(double)*count);
	int sharedBytes = 512*sizeof(double);
	
	hipError_t err = hipGetLastError();
	if (hipSuccess != err){
		printf("CUDA error in ScaLBL_D3Q19_AAeven_Flux_BC_z (memory allocation): %s \n",hipGetErrorString(err));
	}

	// compute the local flux and store the result
	dvc_ScaLBL_D3Q19_AAeven_Flux_BC_z<<<GRID,512,sharedBytes>>>(list, dist, flux, area, dvcsum, count, N);
	err = hipGetLastError();
	if (hipSuccess != err){
		printf("CUDA error in ScaLBL_D3Q19_AAeven_Flux_BC_z (kernel): %s \n",hipGetErrorString(err));
	}

	// Now read the total flux
	hipMemcpy(&sum[0],dvcsum,sizeof(double),hipMemcpyDeviceToHost);
	din=sum[0];
	err = hipGetLastError();
	if (hipSuccess != err){
		printf("CUDA error in ScaLBL_D3Q19_AAeven_Flux_BC_z (reduction): %s \n",hipGetErrorString(err));
	}

	// free the memory needed for reduction
	hipFree(dvcsum);

	return din;
}

extern "C" double ScaLBL_D3Q19_AAodd_Flux_BC_z(int *neighborList, int *list, double *dist, double flux, 
		double area, int count, int N){

	int GRID = count / 512 + 1;

	// IMPORTANT -- this routine may fail if Nx*Ny > 512*512
	if (count > 512*512){
		printf("WARNING (ScaLBL_D3Q19_AAodd_Flux_BC_z): CUDA reduction operation may fail if count > 512*512");
	}

	// Allocate memory to store the sums
	double din;
	double sum[1];
 	double *dvcsum;
	hipMalloc((void **)&dvcsum,sizeof(double)*count);
	hipMemset(dvcsum,0,sizeof(double)*count);
	int sharedBytes = 512*sizeof(double);
	hipError_t err = hipGetLastError();
	if (hipSuccess != err){
		printf("CUDA error in ScaLBL_D3Q19_AAodd_Flux_BC_z (memory allocation): %s \n",hipGetErrorString(err));
	}

	// compute the local flux and store the result
	dvc_ScaLBL_D3Q19_AAodd_Flux_BC_z<<<GRID,512,sharedBytes>>>(neighborList, list, dist, flux, area, dvcsum, count, N);
	err = hipGetLastError();
	if (hipSuccess != err){
		printf("CUDA error in ScaLBL_D3Q19_AAodd_Flux_BC_z (kernel): %s \n",hipGetErrorString(err));
	}
	// Now read the total flux
	hipMemcpy(&sum[0],dvcsum,sizeof(double),hipMemcpyDeviceToHost);
	din=sum[0];
	err = hipGetLastError();
	if (hipSuccess != err){
		printf("CUDA error in ScaLBL_D3Q19_AAodd_Flux_BC_z (reduction): %s \n",hipGetErrorString(err));
	}

	// free the memory needed for reduction
	hipFree(dvcsum);

	return din;
}


extern "C" double deviceReduce(double *in, double* out, int N) {
	int threads = 512;
	int blocks = min((N + threads - 1) / threads, 1024);

	double sum = 0.f;
	deviceReduceKernel<<<blocks, threads>>>(in, out, N);
	deviceReduceKernel<<<1, 1024>>>(out, out, blocks);
	return sum;
}

extern "C" void ScaLBL_D3Q19_Reflection_BC_z(int *list, double *dist, int count, int Np){
	int GRID = count / 512 + 1;
	dvc_ScaLBL_D3Q19_Reflection_BC_z<<<GRID,512>>>(list, dist, count, Np);
	hipError_t err = hipGetLastError();
	if (hipSuccess != err){
		printf("CUDA error in ScaLBL_D3Q19_Reflection_BC_z (kernel): %s \n",hipGetErrorString(err));
	}
}

extern "C" void ScaLBL_D3Q19_Reflection_BC_Z(int *list, double *dist, int count, int Np){
	int GRID = count / 512 + 1;
	dvc_ScaLBL_D3Q19_Reflection_BC_Z<<<GRID,512>>>(list, dist, count, Np);
	hipError_t err = hipGetLastError();
	if (hipSuccess != err){
		printf("CUDA error in ScaLBL_D3Q19_Reflection_BC_Z (kernel): %s \n",hipGetErrorString(err));
	}
}

extern "C" void ScaLBL_D3Q19_AAeven_MRT(double *dist, int start, int finish, int Np, double rlx_setA, double rlx_setB, double Fx,
       double Fy, double Fz){
       
       dvc_ScaLBL_AAeven_MRT<<<NBLOCKS,NTHREADS >>>(dist,start,finish,Np,rlx_setA,rlx_setB,Fx,Fy,Fz);

       hipError_t err = hipGetLastError();
	if (hipSuccess != err){
		printf("CUDA error in ScaLBL_D3Q19_AAeven_MRT: %s \n",hipGetErrorString(err));
	}
}

extern "C" void ScaLBL_D3Q19_AAodd_MRT(int *neighborlist, double *dist, int start, int finish, int Np, double rlx_setA, double rlx_setB, double Fx,
       double Fy, double Fz){
       
       dvc_ScaLBL_AAodd_MRT<<<NBLOCKS,NTHREADS >>>(neighborlist,dist,start,finish,Np,rlx_setA,rlx_setB,Fx,Fy,Fz);

       hipError_t err = hipGetLastError();
	if (hipSuccess != err){
		printf("CUDA error in ScaLBL_D3Q19_AAeven_MRT: %s \n",hipGetErrorString(err));
	}
}

